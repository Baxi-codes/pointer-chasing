#include "hip/hip_runtime.h"
// libraries
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <omp.h>

#include <cutil_inline.h>

#include "cache_kernel.h"

/* ================================================================== */
/* GPU kernel error checking function */
void gpu_check_error__srcpos (FILE* fp, const char* filename, size_t line)
{
  hipError_t C_E = hipGetLastError ();
  if (C_E) {
    fprintf (fp, "*** [%s:%lu] CUDA ERROR %d: %s ***\n", filename, line, C_E, 
						 hipGetErrorString (C_E));
    fflush (fp);
    exit (-1); /* abort program */
  }
}
/* ================================================================== */


/* ================================================================== */
/* Read program input */
void readConfig(int* wordsPerThread, int* nThreads, int* bSize, char** argv)
{
	*wordsPerThread = atoi (argv[1]);
	*nThreads = atoi (argv[2]);
	*bSize = atoi (argv[3]);

}
/* ================================================================== */


/* ================================================================== */
/* Validates the output to make sure the kernel ran correctly */
int validateResults(int nThreads, int bSize, int wordsPerThread, int* out, int* in)
{
	int i, j;
	int tmp, cnt;
	int* test;
	
	test = (int*) malloc (nThreads * sizeof (int));
	for(i = 0; i < nThreads; i++) {
		test[i] = 0;
	}

	for(i = 0; i < nThreads; i++) {
		tmp = in[i % bSize];
		for(j = 0; j < wordsPerThread; j++) {	
			tmp = in[tmp];
		}
		test[i] = tmp;
	}

	cnt = 0;
	for(i = 0; i < nThreads; i++) {
		if(test[i] != out[i]) cnt++;
	}
	
	return cnt;
}
/* ================================================================== */


/* ================================================================== */
/* Used to clear the cache before the main test */
__global__ void clear_cache (int *in)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	in[tid] = in[tid] + 1;
}
/* ================================================================== */


/* ================================================================== */
/* Kernel execution function */
float runTest (int* h_out, int* d_out, int* d_pchase, int nThreads, int bSize, int wordsPerThread)
{
  hipEvent_t start, stop;
	float total_time_taken;

	int num_blocks = (nThreads + bSize - 1) / bSize;

  dim3 grid (num_blocks);
  dim3 threads (bSize);

	fprintf (stderr, "number of iterations is %d\n", NUM_ITER);


  /* Start timer */
	cutilSafeCall (hipEventCreate (&start));
	cutilSafeCall (hipEventCreate (&stop));
	cutilSafeCall (hipEventRecord (start, 0));

		for(int iter = 0; iter < NUM_ITER; iter++) {
		switch (wordsPerThread) {
			case 1:
				cache_kernel_1 <<<grid, threads>>> (nThreads, d_out, d_pchase);	
				break;
			case 2:
				cache_kernel_2 <<<grid, threads>>> (nThreads, d_out, d_pchase);	
				break;
			case 4:
				cache_kernel_4 <<<grid, threads>>> (nThreads, d_out, d_pchase);	
				break;
			case 8:
				cache_kernel_8 <<<grid, threads>>> (nThreads, d_out, d_pchase);	
				break;
			case 16:
				cache_kernel_16 <<<grid, threads>>> (nThreads, d_out, d_pchase);	
				break;
			case 32:
				cache_kernel_32 <<<grid, threads>>> (nThreads, d_out, d_pchase);	
				break;
			case 64:
				cache_kernel_64 <<<grid, threads>>> (nThreads, d_out, d_pchase);	
				break;
			case 128:
				cache_kernel_128 <<<grid, threads>>> (nThreads, d_out, d_pchase);	
				break;
			case 256:
				cache_kernel_256 <<<grid, threads>>> (nThreads, d_out, d_pchase);	
				break;
			case 512:
				cache_kernel_512 <<<grid, threads>>> (nThreads, d_out, d_pchase);	
				break;
			case 1024:
				cache_kernel_1024 <<<grid, threads>>> (nThreads, d_out, d_pchase);	
				break;
			case 2048:
				cache_kernel_2048 <<<grid, threads>>> (nThreads, d_out, d_pchase);	
				break;
			case 4096:
				cache_kernel_4096<<<grid, threads>>> (nThreads, d_out, d_pchase);	
				break;
			default:
				fprintf(stderr, "Invalid wordsPerThread: %d\n", wordsPerThread);
				total_time_taken = -1.0f;
				break;
		}
	}

  /* End timer */
  cutilSafeCall (hipEventRecord (stop, 0));
  cutilSafeCall (hipDeviceSynchronize ());
  cutilSafeCall (hipEventElapsedTime (&total_time_taken, start, stop));

  gpu_check_error__srcpos (stderr, __FILE__, __LINE__);

  /* Copy results back to host */
  cutilSafeCall (hipMemcpy (h_out, d_out, nThreads * sizeof (int),
                             hipMemcpyDeviceToHost));
	return total_time_taken; 
}
/* ================================================================== */


int main(int argc, char** argv)
{
	int i;
	long double total_dram;

	/* Timer */
	float total_time_taken;

	/* Execution parameters */
	int wordsPerThread;
	int nThreads;
	int bSize;

	/* Data structures */
  int* d_out;
  int* h_out;
	int* h_pchase;
	int* d_pchase;


	/* Read program parameters */
  if(argc < 4) {
    fprintf(stderr, "usage: %s <word/thread> <# threads> <threads/block>\n",
						argv[0]);
    exit (0);
  } else {
		readConfig (&wordsPerThread, &nThreads, &bSize, argv);
	}


	/* Find the best GPU in the system */
  hipSetDevice(cutGetMaxGflopsDeviceId ());


	/* Clear the cache before starting the test */
	/* For the shared memory test this isn't really necessary but we do it
		 just in case 
	 */
	{
		int cache_clear = 4 * 1024 * 1024; /* 4 MB */
		int* h_temp = (int*) malloc (cache_clear);
		for(int i = 0; i < cache_clear / sizeof (int); i++) h_temp[i] = 1;
		int* d_temp;
		cutilSafeCall (hipMalloc ((void**) &d_temp, cache_clear));
		cutilSafeCall (hipMemcpy (d_temp, h_temp, cache_clear,
															 hipMemcpyHostToDevice));

		int nt = cache_clear / sizeof (int);
		int nb = nt / 512;
		clear_cache <<<nb, 512>>> (d_temp);
  	cutilSafeCall (hipDeviceSynchronize ());

		free (h_temp);
		cutilSafeCall (hipFree (d_temp));
	}


	/* Allocate memory */
	h_out = (int*) malloc (nThreads * sizeof (int));
	h_pchase = (int*) malloc (bSize * sizeof (int));
	cutilSafeCall (hipMalloc ((void**) &d_out, nThreads * sizeof (int)));
	cutilSafeCall (hipMalloc ((void**) &d_pchase, bSize * sizeof (int)));


	/* Initialize memory */
	for(i = 0; i < bSize - 1; i++) {
		h_pchase[i] = i;
	}
	h_pchase[bSize - 1] = bSize - 1;


	/* Copy memory to device */
	cutilSafeCall (hipMemcpy (d_pchase, h_pchase, bSize * sizeof (int), 
														 hipMemcpyHostToDevice));


	/* Execute the kernel */
	total_time_taken = runTest (h_out, d_out, d_pchase, nThreads, bSize, 
															wordsPerThread);


	/* Validate results */
	/* This will take some time so you may want to turn it off when doing
		 multiple tests */
	fprintf(stderr, "Results validated: %d errors\n", validateResults (nThreads, 
																															bSize, 
																															wordsPerThread, 
																															h_out, h_pchase));


	/* Print performance statistics */
	total_time_taken = total_time_taken / NUM_ITER;
	total_dram = ((wordsPerThread + 1) * (1.0 * nThreads/1e9)) * sizeof (int);
  fprintf (stderr, "Time taken to load %Lg GBs: %f (ms)\n", total_dram, 
					 total_time_taken);
	fprintf (stderr, "Effective bandwidth: %Lg (GB/s)\n", 
					 (total_dram/total_time_taken * 1e3));


	/* Free memory */
	free (h_out);
	cutilSafeCall (hipFree (d_out));
	free (h_pchase);
	cutilSafeCall (hipFree (d_pchase));

	return 0;
}
