#include "hip/hip_runtime.h"
/*
MIT License
Copyright 2020 Jee W. Choi, Marat Dukhan, and Xing Liu
Permission is hereby granted, free of charge, to any person obtaining a copy of 
this software and associated documentation files (the "Software"), to deal in 
the Software without restriction, including without limitation the rights to use, 
copy, modify, merge, publish, distribute, sublicense, and/or sell copies of the 
Software, and to permit persons to whom the Software is furnished to do so, subject 
to the following conditions:
The above copyright notice and this permission notice shall be included in all 
copies or substantial portions of the Software.
THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR IMPLIED, 
INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS FOR A 
PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT 
HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF 
CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE 
OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
*/

// libraries
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <omp.h>
#include <stdint.h>

#include <cutil_inline.h>

/* ================================================================== */
/* GPU memory random access benchmark */
__global__ void gpu_rand (int nThreads, int arraySize, int* p_chase, int* d_result, int nIter)
{
  int i;
  int gid = threadIdx.x + blockIdx.x * blockDim.x;
  unsigned int temp;

  temp = p_chase[gid];
  for(i = 0; i < nIter; i++) {
    temp = p_chase[temp];
  }

  d_result[gid] = temp;
}
/* ================================================================== */


/* ================================================================== */
/* Pointer chasing array generator */
double Log2 (double n)
{
  return log(n) / log (2);
}

void findShifter (int nBits, unsigned int shifter[3])
{
	switch (nBits) {
		case 30:
			 shifter[0] = 1;
			 shifter[1] = 11;
			 shifter[2] = 24;
			 break;
		 case 29:
			 shifter[0] = 2;
			 shifter[1] = 4;
			 shifter[2] = 5;
			 break;
		 case 28:
			 shifter[0] = 1;
			 shifter[1] = 3;
			 shifter[2] = 9;
			 break;
		 case 27:
			 shifter[0] = 1;
			 shifter[1] = 2;
			 shifter[2] = 6;
			 break;
		 case 26:
			 shifter[0] = 2;
			 shifter[1] = 5;
			 shifter[2] = 3;
			 break;
		 case 25:
			 shifter[0] = 2;
			 shifter[1] = 1;
			 shifter[2] = 6;
			 break;
		 case 24:
			 shifter[0] = 6;
			 shifter[1] = 1;
			 shifter[2] = 9;
			 break;
		 case 23:
			 shifter[0] = 7;
			 shifter[1] = 8;
			 shifter[2] = 2;
			 break;
		 case 22:
			 shifter[0] = 5;
			 shifter[1] = 3;
			 shifter[2] = 2;
			 break;
		 case 21:
			 shifter[0] = 4;
			 shifter[1] = 3;
			 shifter[2] = 6;
			 break;
		 case 20:
			 shifter[0] = 2;
			 shifter[1] = 1;
			 shifter[2] = 9;
			 break;
		 case 19:
			 shifter[0] = 9;
			 shifter[1] = 6;
			 shifter[2] = 4;
			 break;
		 case 18:
			 shifter[0] = 3;
			 shifter[1] = 9;
			 shifter[2] = 8;
			 break;
		 case 17:
			 shifter[0] = 9;
			 shifter[1] = 6;
			 shifter[0] = 1;
			 break;
		 case 16:
			 shifter[0] = 8;
			 shifter[1] = 9;
			 shifter[2] = 5;
			 break;
		 case 15:
			 shifter[0] = 4;
			 shifter[1] = 1;
			 shifter[2] = 9;
			 break;
		 case 14:
			 shifter[0] = 9;
			 shifter[1] = 7;
			 shifter[2] = 2;
			 break;
		 case 13:
			 shifter[0] = 8;
			 shifter[1] = 5;
			 shifter[2] = 2;
			 break;
		 case 12:
			 shifter[0] = 7;
			 shifter[1] = 5;
			 shifter[2] = 9;
			 break;
		 case 11:
			 shifter[0] = 6;
			 shifter[1] = 4;
			 shifter[2] = 7;
			 break;
		 case 10:
			 shifter[0] = 5;
			 shifter[1] = 1;
			 shifter[2] = 3;
			 break;
		 case 9:
			 shifter[0] = 8;
			 shifter[1] = 4;
			 shifter[2] = 5;
			 break;
		 case 8:
			 shifter[0] = 3;
			 shifter[1] = 1;
			 shifter[2] = 5;
			 break;
		 case 7:
			 shifter[0] = 5;
			 shifter[1] = 2;
			 shifter[2] = 6;
			 break;
		 case 6:
			 shifter[0] = 2;
			 shifter[1] = 3;
			 shifter[2] = 1;
			 break;
		 case 5:
			 shifter[0] = 4;
			 shifter[1] = 3;
			 shifter[2] = 1;
			 break;
		 case 4:
			 shifter[0] = 3;
			 shifter[1] = 1;
			shifter[2] = 2;
			 break;
	}
}
/* ================================================================== */


/* ================================================================== */
/* GPU kernel error checking function */
void gpu_check_error__srcpos (FILE* fp, const char* filename, size_t line)
{
  hipError_t C_E = hipGetLastError ();
  if (C_E) {
		 fprintf (fp, "*** [%s:%lu] CUDA ERROR %d: %s ***\n", filename, line, C_E,
					     hipGetErrorString (C_E));
		 fflush (fp);
		 exit (-1); /* abort program */
  }
}
/* ================================================================== */


/* ================================================================== */
/* Read program input */
void readConfig(int* nThreads, int* bSize, int* arraySize, int* nIter, 
								char** argv)
{
	*nThreads = atoi (argv[1]);	
	*bSize = atoi (argv[2]);	
	*arraySize = atoi (argv[3]);
	*nIter = atoi (argv[4]);
}
/* ================================================================== */


/* ================================================================== */
/* Validate results */
int validateResults (int nThreads, int nIter, int* h_Input, int* h_Output)
{
	unsigned int i, j, cnt, temp;

	cnt = 0;
	for(i = 0; i < nThreads; i++) {
		temp = h_Input[i];
		for(j = 0; j < nIter; j++) {
			temp = h_Input[temp];
		}
		if (temp != h_Output[i]) cnt++;
	}

	return cnt;
}
/* ================================================================== */


int main(int argc, char** argv)
{	
	int i;


	/* Timer */
	hipEvent_t start, stop;
	float total_time;


  /* Execution parameters */
  int nThreads;
  int bSize;
  int arraySize;
	int nIter;

	int num_blocks;


	/* Memory */
	int* h_array;
	int* h_result;
	int* p_chase;
	int* d_result;


	/* Read input parameters */
	if(argc != 5) {
		fprintf(stderr, "usage: %s <# threads> <TB size> <array size> <# iter>\n", 
						argv[0]);
		exit (0);
	} else {
		readConfig (&nThreads, &bSize, &arraySize, &nIter, argv);
	}
	num_blocks = (nThreads + bSize - 1) / bSize;


	/* Find the best GPU in the system */
  fprintf(stderr, "Running on device: %d\n", cutGetMaxGflopsDeviceId ());
	hipSetDevice(cutGetMaxGflopsDeviceId ());


	/* Allocate memory */
	uint32_t nBits = Log2 ((double) arraySize);	
	uint32_t shifter[3];
	findShifter (nBits, shifter);

	h_array = (int*) malloc (arraySize * sizeof (TYPE));
	cutilSafeCall (hipMalloc ((void**) &p_chase, arraySize * sizeof (TYPE)));
	cutilSafeCall (hipMalloc ((void**) &d_result, nThreads * sizeof (TYPE)));


	/* Initialize memory on CPU and GPU */
	h_array[0] = 1;
	uint32_t index = 1;
	for(i = 0; i < arraySize - 2; i++) {
		uint32_t temp = index;
		index ^= index >> shifter[0];
		index ^= index << shifter[1];
		index &= ((1u << nBits) - 1u);
		index ^= index >> shifter[2];
		h_array[temp] = index;
	}
	cutilSafeCall (hipMemcpy (p_chase, h_array, arraySize * sizeof (TYPE),
														 hipMemcpyHostToDevice));


  dim3 grid (num_blocks);
  dim3 threads (bSize);

	total_time = 0.0;

	#pragma omp parallel num_threads(2)
	{
		#pragma omp sections
		{
			#pragma omp section
			/* Power measurement thread */
			{
				fprintf (stderr, "Starting power measuremnt\n");
        fprintf (stderr, "Power measurement code running on thread %d\n",
                 omp_get_thread_num ());
			}
			#pragma omp section
			/* Kernel execution thread */
			{
			  /* Start timer */
				cutilSafeCall (hipEventCreate (&start));
				cutilSafeCall (hipEventCreate (&stop));
				cutilSafeCall (hipEventRecord (start, 0));

				gpu_rand <<<grid, threads>>> (nThreads, arraySize, p_chase, d_result, 
																			nIter);

				/* End timer */
				cutilSafeCall (hipEventRecord (stop, 0));
				cutilSafeCall (hipDeviceSynchronize ());
				cutilSafeCall (hipEventElapsedTime (&total_time, start, stop));

        fprintf (stderr, "Test kernel running on thread %d\n",
                 omp_get_thread_num ());
			}
		}
	}
	

	/* Check for kernel execution errors */
  gpu_check_error__srcpos (stderr, __FILE__, __LINE__);


	/* Compute performance statistics */
	double data = (1.0 * nThreads * (1 + nIter)) / 1e9;
	fprintf (stderr, "Data (GB): %lf ::: Time (ms): %f\n", data, total_time);
	fprintf (stderr, "Time taken: %f (s)\n", total_time/1e3);
	fprintf (stderr, "Performance: %f (GA/s)\n", data / (total_time/1e3));


  /* Copy results back to host */
	h_result = (int*) malloc (nThreads * sizeof (TYPE));
  cutilSafeCall (hipMemcpy (h_result, d_result, nThreads * sizeof (TYPE),
                             hipMemcpyDeviceToHost));


	/* Validate results */
	fprintf (stderr, "Results validated: %d\n", validateResults (nThreads, nIter,
																															 h_array, 
																															 h_result));


	/* Free memory */
  free (h_array);
  free (h_result);
  cutilSafeCall (hipFree (p_chase));
  cutilSafeCall (hipFree (d_result));


	return 0;
}
