/*
MIT License
Copyright 2020 Jee W. Choi, Marat Dukhan, and Xing Liu
Permission is hereby granted, free of charge, to any person obtaining a copy of 
this software and associated documentation files (the "Software"), to deal in 
the Software without restriction, including without limitation the rights to use, 
copy, modify, merge, publish, distribute, sublicense, and/or sell copies of the 
Software, and to permit persons to whom the Software is furnished to do so, subject 
to the following conditions:
The above copyright notice and this permission notice shall be included in all 
copies or substantial portions of the Software.
THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR IMPLIED, 
INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS FOR A 
PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT 
HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF 
CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE 
OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
*/

// libraries
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>

#include <cutil_inline.h>

// #include "intensity_kernel.h"
#include "py.intensity_kernel.h"

#define NUM_ITER 10

// GPU kernel error checking function
void gpu_check_error__srcpos (FILE* fp, const char* filename, size_t line)
{
  hipError_t C_E = hipGetLastError ();
  if (C_E) {
    fprintf (fp, "*** [%s:%lu] CUDA ERROR %d: %s ***\n", filename, line, C_E,
             hipGetErrorString (C_E));
    fflush (fp);
    exit (-1); /* abort program */
  }
}


void readConfig(FILE *fp, int* num_threads, int* block_size,
                int* streams_per_thread)
{
	char line[20];
	if(fgets(line, 20, fp) != NULL) {
		*num_threads = atoi (line);
	} else {
		fprintf(stderr, "Can't find number of threads in the config file\n");
		exit(0);
	}
	if(fgets(line, 20, fp) != NULL) {
		*block_size = atoi (line);
	} else {
		fprintf(stderr, "Can't find block size in the config file\n");
		exit(0);
	}

	if(fgets(line, 20, fp) != NULL) {
		*streams_per_thread = atoi (line);
	} else {
		fprintf(stderr, "Can't find streams per thread in the config file\n");
		exit(0);
	}
}

void executeKernel(int num_blocks, int bSize, int nThreads, int nStreams, 
									 int nFlops, TYPE* d_in, TYPE* d_out)
{
	dim3 grid (num_blocks);
	dim3 threads (bSize);

	/*
	fprintf(stderr, "Executing kernel %d nStreams and %d nFlops\n", 
					nStreams, nFlops);
	 */

  switch (nStreams) {
    case 1:
			switch (nFlops) {
				case 1:
		      intensity_kernel_1_1 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 2:
		      intensity_kernel_1_2 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 3:
		      intensity_kernel_1_3 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 4:
		      intensity_kernel_1_4 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 5:
		      intensity_kernel_1_5 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 6:
		      intensity_kernel_1_6 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 7:
		      intensity_kernel_1_7 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 8:
		      intensity_kernel_1_8 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 9:
		      intensity_kernel_1_8 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 10:
		      intensity_kernel_1_10 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 11:
		      intensity_kernel_1_11 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 12:
		      intensity_kernel_1_12 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 13:
		      intensity_kernel_1_13 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 14:
		      intensity_kernel_1_14 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 15:
		      intensity_kernel_1_15 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 16:
		      intensity_kernel_1_16 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 17:
		      intensity_kernel_1_17 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 18:
		      intensity_kernel_1_18 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 19:
		      intensity_kernel_1_19 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 20:
		      intensity_kernel_1_20 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 21:
		      intensity_kernel_1_21 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 22:
		      intensity_kernel_1_22 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 23:
		      intensity_kernel_1_23 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 24:
		      intensity_kernel_1_24 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 25:
		      intensity_kernel_1_25 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 26:
		      intensity_kernel_1_26 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 27:
		      intensity_kernel_1_27 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 28:
		      intensity_kernel_1_28 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 29:
		      intensity_kernel_1_29 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 30:
		      intensity_kernel_1_30 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 31:
		      intensity_kernel_1_31 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 32:
		      intensity_kernel_1_32 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 64:
		      intensity_kernel_1_64 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 128:
		      intensity_kernel_1_128 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 256:
		      intensity_kernel_1_256 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 512:
		      intensity_kernel_1_512 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				default:
					fprintf(stderr, "Invalid combination: %d nStreams and %d nFlops\n",
									nStreams, nFlops);
					break;
			}
      break;
    case 2:
			switch (nFlops) {
				case 1:
		      intensity_kernel_2_1 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 2:
		      intensity_kernel_2_2 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 3:
		      intensity_kernel_2_3 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 4:
		      intensity_kernel_2_4 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 5:
		      intensity_kernel_2_5 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 6:
		      intensity_kernel_2_6 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 7:
		      intensity_kernel_2_7 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 8:
		      intensity_kernel_2_8 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 9:
		      intensity_kernel_2_8 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 10:
		      intensity_kernel_2_10 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 11:
		      intensity_kernel_2_11 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 12:
		      intensity_kernel_2_12 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 13:
		      intensity_kernel_2_13 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 14:
		      intensity_kernel_2_14 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 15:
		      intensity_kernel_2_15 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 16:
		      intensity_kernel_2_16 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 17:
		      intensity_kernel_2_17 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 18:
		      intensity_kernel_2_18 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 19:
		      intensity_kernel_2_19 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 20:
		      intensity_kernel_2_20 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 21:
		      intensity_kernel_2_21 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 22:
		      intensity_kernel_2_22 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 23:
		      intensity_kernel_2_23 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 24:
		      intensity_kernel_2_24 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 25:
		      intensity_kernel_2_25 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 26:
		      intensity_kernel_2_26 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 27:
		      intensity_kernel_2_27 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 28:
		      intensity_kernel_2_28 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 29:
		      intensity_kernel_2_29 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 30:
		      intensity_kernel_2_30 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 31:
		      intensity_kernel_2_31 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 32:
		      intensity_kernel_2_32 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 64:
		      intensity_kernel_2_64 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 128:
		      intensity_kernel_2_128 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 256:
		      intensity_kernel_2_256 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 512:
		      intensity_kernel_2_512 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 1024:
		      intensity_kernel_2_1024 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 2048:
		      intensity_kernel_2_2048 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 4096:
		      intensity_kernel_2_4096 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				default:
					fprintf(stderr, "Invalid combination: %d nStreams and %d nFlops\n",
									nStreams, nFlops);
					break;
			}
      break;
    case 4:
			switch (nFlops) {
				case 1:
		      intensity_kernel_4_1 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 2:
		      intensity_kernel_4_2 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 3:
		      intensity_kernel_4_3 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 4:
		      intensity_kernel_4_4 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 5:
		      intensity_kernel_4_5 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 6:
		      intensity_kernel_4_6 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 7:
		      intensity_kernel_4_7 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 8:
		      intensity_kernel_4_8 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 9:
		      intensity_kernel_4_8 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 10:
		      intensity_kernel_4_10 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 11:
		      intensity_kernel_4_11 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 12:
		      intensity_kernel_4_12 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 13:
		      intensity_kernel_4_13 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 14:
		      intensity_kernel_4_14 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 15:
		      intensity_kernel_4_15 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 16:
		      intensity_kernel_4_16 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 17:
		      intensity_kernel_4_17 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 18:
		      intensity_kernel_4_18 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 19:
		      intensity_kernel_4_19 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 20:
		      intensity_kernel_4_20 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 21:
		      intensity_kernel_4_21 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 22:
		      intensity_kernel_4_22 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 23:
		      intensity_kernel_4_23 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 24:
		      intensity_kernel_4_24 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 25:
		      intensity_kernel_4_25 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 26:
		      intensity_kernel_4_26 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 27:
		      intensity_kernel_4_27 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 28:
		      intensity_kernel_4_28 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 29:
		      intensity_kernel_4_29 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 30:
		      intensity_kernel_4_30 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 31:
		      intensity_kernel_4_31 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 32:
		      intensity_kernel_4_32 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 64:
		      intensity_kernel_4_64 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 128:
		      intensity_kernel_4_128 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				default:
					fprintf(stderr, "Invalid combination: %d nStreams and %d nFlops\n",
									nStreams, nFlops);
					break;
			}
      break;
    case 8:
			switch (nFlops) {
				case 1:
		      intensity_kernel_8_1 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 2:
		      intensity_kernel_8_2 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 3:
		      intensity_kernel_8_3 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 4:
		      intensity_kernel_8_4 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 5:
		      intensity_kernel_8_5 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 6:
		      intensity_kernel_8_6 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 7:
		      intensity_kernel_8_7 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 8:
		      intensity_kernel_8_8 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 9:
		      intensity_kernel_8_8 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 10:
		      intensity_kernel_8_10 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 11:
		      intensity_kernel_8_11 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 12:
		      intensity_kernel_8_12 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 13:
		      intensity_kernel_8_13 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 14:
		      intensity_kernel_8_14 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 15:
		      intensity_kernel_8_15 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 16:
		      intensity_kernel_8_16 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 17:
		      intensity_kernel_8_17 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 18:
		      intensity_kernel_8_18 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 19:
		      intensity_kernel_8_19 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 20:
		      intensity_kernel_8_20 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 21:
		      intensity_kernel_8_21 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 22:
		      intensity_kernel_8_22 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 23:
		      intensity_kernel_8_23 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 24:
		      intensity_kernel_8_24 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 25:
		      intensity_kernel_8_25 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 26:
		      intensity_kernel_8_26 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 27:
		      intensity_kernel_8_27 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 28:
		      intensity_kernel_8_28 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 29:
		      intensity_kernel_8_29 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 30:
		      intensity_kernel_8_30 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 31:
		      intensity_kernel_8_31 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 32:
		      intensity_kernel_8_32 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 48:
		      intensity_kernel_8_48 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 64:
		      intensity_kernel_8_64 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 128:
		      intensity_kernel_8_128 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 160:
		      intensity_kernel_8_160 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 192:
		      intensity_kernel_8_192 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				default:
					fprintf(stderr, "Invalid combination: %d nStreams and %d nFlops\n",
									nStreams, nFlops);
					break;
			}
      break;
    case 16:
			switch (nFlops) {
				case 1:
		      intensity_kernel_16_1 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 2:
		      intensity_kernel_16_2 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 3:
		      intensity_kernel_16_3 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 4:
		      intensity_kernel_16_4 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 5:
		      intensity_kernel_16_5 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 6:
		      intensity_kernel_16_6 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 7:
		      intensity_kernel_16_7 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 8:
		      intensity_kernel_16_8 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 9:
		      intensity_kernel_16_8 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 10:
		      intensity_kernel_16_10 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 11:
		      intensity_kernel_16_11 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 12:
		      intensity_kernel_16_12 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 13:
		      intensity_kernel_16_13 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 14:
		      intensity_kernel_16_14 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 15:
		      intensity_kernel_16_15 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 16:
		      intensity_kernel_16_16 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 17:
		      intensity_kernel_16_17 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 18:
		      intensity_kernel_16_18 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 19:
		      intensity_kernel_16_19 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 20:
		      intensity_kernel_16_20 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 21:
		      intensity_kernel_16_21 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 22:
		      intensity_kernel_16_22 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 23:
		      intensity_kernel_16_23 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 24:
		      intensity_kernel_16_24 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 25:
		      intensity_kernel_16_25 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 26:
		      intensity_kernel_16_26 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 27:
		      intensity_kernel_16_27 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 28:
		      intensity_kernel_16_28 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 29:
		      intensity_kernel_16_29 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 30:
		      intensity_kernel_16_30 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 31:
		      intensity_kernel_16_31 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 32:
		      intensity_kernel_16_32 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				default:
					fprintf(stderr, "Invalid combination: %d nStreams and %d nFlops\n",
									nStreams, nFlops);
					break;
			}
      break;
    case 32:
			switch (nFlops) {
				case 1:
		      intensity_kernel_32_1 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 2:
		      intensity_kernel_32_2 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 3:
		      intensity_kernel_32_3 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 4:
		      intensity_kernel_32_4 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 5:
		      intensity_kernel_32_5 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 6:
		      intensity_kernel_32_6 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 7:
		      intensity_kernel_32_7 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 8:
		      intensity_kernel_32_8 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 9:
		      intensity_kernel_32_8 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 10:
		      intensity_kernel_32_10 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 11:
		      intensity_kernel_32_11 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 12:
		      intensity_kernel_32_12 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 13:
		      intensity_kernel_32_13 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 14:
		      intensity_kernel_32_14 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 15:
		      intensity_kernel_32_15 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 16:
		      intensity_kernel_32_16 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 17:
		      intensity_kernel_32_17 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 18:
		      intensity_kernel_32_18 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 19:
		      intensity_kernel_32_19 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 20:
		      intensity_kernel_32_20 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 21:
		      intensity_kernel_32_21 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 22:
		      intensity_kernel_32_22 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 23:
		      intensity_kernel_32_23 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 24:
		      intensity_kernel_32_24 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 25:
		      intensity_kernel_32_25 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 26:
		      intensity_kernel_32_26 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 27:
		      intensity_kernel_32_27 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 28:
		      intensity_kernel_32_28 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 29:
		      intensity_kernel_32_29 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 30:
		      intensity_kernel_32_30 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 31:
		      intensity_kernel_32_31 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 32:
		      intensity_kernel_32_32 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				default:
					fprintf(stderr, "Invalid combination: %d nStreams and %d nFlops\n",
									nStreams, nFlops);
					break;
			}
      break;
    case 64:
			switch (nFlops) {
				case 1:
		      intensity_kernel_64_1 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 2:
		      intensity_kernel_64_2 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 3:
		      intensity_kernel_64_3 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 4:
		      intensity_kernel_64_4 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 5:
		      intensity_kernel_64_5 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 6:
		      intensity_kernel_64_6 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 7:
		      intensity_kernel_64_7 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 8:
		      intensity_kernel_64_8 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 9:
		      intensity_kernel_64_8 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 10:
		      intensity_kernel_64_10 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 11:
		      intensity_kernel_64_11 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 12:
		      intensity_kernel_64_12 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 13:
		      intensity_kernel_64_13 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 14:
		      intensity_kernel_64_14 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 15:
		      intensity_kernel_64_15 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 16:
		      intensity_kernel_64_16 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 17:
		      intensity_kernel_64_17 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 18:
		      intensity_kernel_64_18 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 19:
		      intensity_kernel_64_19 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 20:
		      intensity_kernel_64_20 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 21:
		      intensity_kernel_64_21 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 22:
		      intensity_kernel_64_22 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 23:
		      intensity_kernel_64_23 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 24:
		      intensity_kernel_64_24 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 25:
		      intensity_kernel_64_25 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 26:
		      intensity_kernel_64_26 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 27:
		      intensity_kernel_64_27 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 28:
		      intensity_kernel_64_28 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 29:
		      intensity_kernel_64_29 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 30:
		      intensity_kernel_64_30 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 31:
		      intensity_kernel_64_31 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 32:
		      intensity_kernel_64_32 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				default:
					fprintf(stderr, "Invalid combination: %d nStreams and %d nFlops\n",
									nStreams, nFlops);
					break;
			}
      break;
    case 128:
			switch (nFlops) {
				case 1:
		      intensity_kernel_128_1 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 2:
		      intensity_kernel_128_2 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 3:
		      intensity_kernel_128_3 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 4:
		      intensity_kernel_128_4 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 5:
		      intensity_kernel_128_5 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 6:
		      intensity_kernel_128_6 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 7:
		      intensity_kernel_128_7 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 8:
		      intensity_kernel_128_8 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 9:
		      intensity_kernel_128_8 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 10:
		      intensity_kernel_128_10 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 11:
		      intensity_kernel_128_11 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 12:
		      intensity_kernel_128_12 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 13:
		      intensity_kernel_128_13 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 14:
		      intensity_kernel_128_14 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 15:
		      intensity_kernel_128_15 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 16:
		      intensity_kernel_128_16 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 17:
		      intensity_kernel_128_17 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 18:
		      intensity_kernel_128_18 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 19:
		      intensity_kernel_128_19 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 20:
		      intensity_kernel_128_20 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 21:
		      intensity_kernel_128_21 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 22:
		      intensity_kernel_128_22 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 23:
		      intensity_kernel_128_23 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 24:
		      intensity_kernel_128_24 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 25:
		      intensity_kernel_128_25 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 26:
		      intensity_kernel_128_26 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 27:
		      intensity_kernel_128_27 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 28:
		      intensity_kernel_128_28 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 29:
		      intensity_kernel_128_29 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 30:
		      intensity_kernel_128_30 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 31:
		      intensity_kernel_128_31 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				case 32:
		      intensity_kernel_128_32 <<<grid, threads>>> (nThreads, nStreams,
		                                              	d_in, d_out);
					break;
				default:
					fprintf(stderr, "Invalid combination: %d nStreams and %d nFlops\n",
									nStreams, nFlops);
					break;
			}
      break;
    default:
      fprintf(stderr, "Invalid nStreams: %d\n", nStreams);
      break;
	}
}

int main(int argc, char** argv)
{	
	int i;

	// timer
	float total_time_taken;
	hipEvent_t start, stop;

  // execution parameters
  double intensity;
  int nThreads;
  int bSize;
  int nStreams;
	int nFlops;

	int num_blocks;


	// memory
	TYPE* h_in;
	TYPE* h_out;
	TYPE* d_in;
	TYPE* d_out;

	// file
	FILE* fp;

	if(argc != 3) {
		fprintf(stderr, "usage: %s <intensity> <file name>\n", argv[0]);
		exit (0);
	}

	// intensity is flop-to-byte ratio
	intensity = atof (argv[1]);
	nFlops = intensity * sizeof (TYPE);
	fp = fopen (argv[2], "r");

  if(fp == NULL) {
    fprintf(stderr, "File %s does not exist, exiting...\n", argv[2]);
		exit (0);
  } else {
    fprintf(stderr, "Reading parameters...\n");
    readConfig (fp, &nThreads, &bSize, &nStreams);
    fclose (fp);
  }

	fprintf(stderr,
					"Best bandwidth at %d threads %d block size %d streams/thread\n",
					nThreads, bSize, nStreams);
	fprintf(stderr, "Required flops for intensity of %.2f: %d\n", intensity, 
					nFlops);

	// find the best GPU in the system
  fprintf(stderr, "Running on device: %d\n", cutGetMaxGflopsDeviceId ());
	hipSetDevice(cutGetMaxGflopsDeviceId ());

	num_blocks = (nThreads + bSize - 1) / bSize;

	// allocate memory
	h_in = (TYPE*) malloc (nThreads * nStreams * sizeof (TYPE));
	h_out = (TYPE*) malloc (nThreads * nStreams * sizeof (TYPE));
	cutilSafeCall (hipMalloc ((void**) &d_in, nThreads * nStreams *
														 sizeof (TYPE)));
	cutilSafeCall (hipMalloc ((void**) &d_out, nThreads * nStreams *
														 sizeof (TYPE)));

	// initialize memory
	for(i = 0; i < nThreads * nStreams; i++) {
		h_in[i] = (TYPE) drand48 ();
		h_out[i] = 0.0;
	}
	cutilSafeCall (hipMemcpy (d_in, h_in, nThreads * nStreams * sizeof (TYPE),
														 hipMemcpyHostToDevice));
	cutilSafeCall (hipMemcpy (d_out, h_out, nThreads * nStreams * sizeof (TYPE),
														 hipMemcpyHostToDevice));

	// dim3 grid (num_blocks);
	// dim3 threads (bSize);

	fprintf (stderr, "Sleeping...\n");
	sleep (1);

  // start timer
	cutilSafeCall (hipEventCreate (&start));
	cutilSafeCall (hipEventCreate (&stop));
	cutilSafeCall (hipEventRecord (start, 0));

	for(int iter = 0; iter < NUM_ITER; iter++) {
		executeKernel (num_blocks, bSize, nThreads, nStreams, nFlops, d_in, d_out);
	}

  // end timer
  cutilSafeCall (hipEventRecord (stop, 0));
  // cutilSafeCall (cutilDeviceSynchronize ());
  cutilSafeCall (hipDeviceSynchronize ());
  cutilSafeCall (hipEventElapsedTime (&total_time_taken, start, stop));

	total_time_taken = total_time_taken / NUM_ITER;
  gpu_check_error__srcpos (stderr, __FILE__, __LINE__);

	sleep (2);
  // copy results back
  cutilSafeCall (hipMemcpy (h_out, d_out, nThreads * nStreams * sizeof (TYPE),
                             hipMemcpyDeviceToHost));

  printf("Time taken to trasnfer %f GB and execute %f Gflops: %f (ms)\n", 
				 (((2.0 * nThreads / 1e6) * nStreams * sizeof (TYPE) ) / 1e3), 
				 (((2.0 * nThreads / 1e6) * nStreams * nFlops) / 1e3),
				 total_time_taken);
  printf("Effective performance: %f (GB/s)\n", (((2.0 * nThreads / 1e6) *
         nStreams * sizeof (TYPE) )) / total_time_taken);
  fprintf(stderr, "Effective performance: %f (GFlops/s)\n",
         (((2.0 * nThreads / 1e6) * nFlops * nStreams)) / total_time_taken);


  free (h_in);
  free (h_out);
  cutilSafeCall (hipFree (d_in));
  cutilSafeCall (hipFree (d_out));

	fprintf(stderr, "Target intensity is %.2f\n", intensity);
	return 0;
}
