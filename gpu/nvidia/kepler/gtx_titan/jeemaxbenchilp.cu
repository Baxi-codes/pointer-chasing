#include "hip/hip_runtime.h"
// libraries
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <omp.h>

#include <cutil_inline.h>

#include "py.intensity_kernel.ilp.h"

// GPU kernel error checking function
void gpu_check_error__srcpos (FILE* fp, const char* filename, size_t line)
{
  hipError_t C_E = hipGetLastError ();
  if (C_E) {
    fprintf (fp, "*** [%s:%lu] CUDA ERROR %d: %s ***\n", filename, line, C_E,
             hipGetErrorString (C_E));
    fflush (fp);
    exit (-1); /* abort program */
  }
}


void readConfig(FILE *fp, int* num_threads, int* block_size,
                int* streams_per_thread)
{
	char line[20];
	if(fgets(line, 20, fp) != NULL) {
		*num_threads = atoi (line);
	} else {
		fprintf(stderr, "Can't find number of threads in the config file\n");
		exit(0);
	}
	if(fgets(line, 20, fp) != NULL) {
		*block_size = atoi (line);
	} else {
		fprintf(stderr, "Can't find block size in the config file\n");
		exit(0);
	}

	if(fgets(line, 20, fp) != NULL) {
		*streams_per_thread = atoi (line);
	} else {
		fprintf(stderr, "Can't find streams per thread in the config file\n");
		exit(0);
	}
}


float executeKernel(int num_blocks, int bSize, int nThreads, int nStreams, 
									 int nFlops, TYPE* d_in, TYPE* d_out)
{

  hipEvent_t start, stop;
  float total_time_taken;

	dim3 grid (num_blocks);
	dim3 threads (bSize);

	/*
	fprintf(stderr, "Executing kernel %d nStreams and %d nFlops\n", 
					nStreams, nFlops);
	 */

  cutilSafeCall (hipEventCreate (&start));
  cutilSafeCall (hipEventCreate (&stop));
  cutilSafeCall (hipEventRecord (start, 0));

	nThreads = nThreads * nStreams;
	switch (nFlops) {
		case 1:
			/* intensity 0.25 */
			intensity_kernel_1_1 <<<grid, threads>>> (nThreads, nStreams,
																								d_in, d_out);
			break;
		case 2:
			/* intensity 0.5 */
			intensity_kernel_1_2 <<<grid, threads>>> (nThreads, nStreams,
		                                            d_in, d_out);
			break;
		case 3:
			/* intensity 0.75 */
			intensity_kernel_1_3 <<<grid, threads>>> (nThreads, nStreams,
		                                            d_in, d_out);
			break;

		case 4:
			/* intensity 1.0 */
			intensity_kernel_1_4 <<<grid, threads>>> (nThreads, nStreams,
		                                            d_in, d_out);
			break;
		case 5:
			/* intensity 1.25 */
			intensity_kernel_1_5 <<<grid, threads>>> (nThreads, nStreams,
		                                            d_in, d_out);
			break;
		case 6:
			/* intensity 1.5 */
			intensity_kernel_1_6 <<<grid, threads>>> (nThreads, nStreams,
		                                            d_in, d_out);
			break;
		case 7:
			/* intensity 1.75 */
			intensity_kernel_1_7 <<<grid, threads>>> (nThreads, nStreams,
		                                            d_in, d_out);
			break;
		case 8:
			/* intensity 2.0 */
			intensity_kernel_1_8 <<<grid, threads>>> (nThreads, nStreams,
		                                            d_in, d_out);
			break;
		case 10:
			/* intensity 2.5 */
			intensity_kernel_1_10 <<<grid, threads>>> (nThreads, nStreams,
		                                            d_in, d_out);
			break;
		case 12:
			/* intensity 3.0 */
			intensity_kernel_1_12 <<<grid, threads>>> (nThreads, nStreams,
		                                            d_in, d_out);
			break;
		case 14:
			/* intensity 3.5 */
			intensity_kernel_1_14 <<<grid, threads>>> (nThreads, nStreams,
		                                            d_in, d_out);
			break;
		case 16:
			/* intensity 4.0 */
			intensity_kernel_1_16 <<<grid, threads>>> (nThreads, nStreams,
		                                            d_in, d_out);
			break;
		case 20:
			/* intensity 5.0 */
			intensity_kernel_1_20 <<<grid, threads>>> (nThreads, nStreams,
		                                            d_in, d_out);
			break;
		case 24:
			/* intensity 6.0 */
			intensity_kernel_1_24 <<<grid, threads>>> (nThreads, nStreams,
		                                            d_in, d_out);
			break;
		case 28:
			/* intensity 7.0 */
			intensity_kernel_1_28 <<<grid, threads>>> (nThreads, nStreams,
		                                            d_in, d_out);
			break;
		case 32:
			/* intensity 8.0 */
			intensity_kernel_1_32 <<<grid, threads>>> (nThreads, nStreams,
		                                            d_in, d_out);
			break;
		case 40:
			/* intensity 10.0 */
			intensity_kernel_1_40 <<<grid, threads>>> (nThreads, nStreams,
		                                            d_in, d_out);
			break;
		case 48:
			/* intensity 12.0 */
			intensity_kernel_1_48 <<<grid, threads>>> (nThreads, nStreams,
		                                            d_in, d_out);
			break;
		case 56:
			/* intensity 14.0 */
			intensity_kernel_1_56 <<<grid, threads>>> (nThreads, nStreams,
		                                            d_in, d_out);
			break;
		case 64:
			/* intensity 16.0 */
			intensity_kernel_1_64 <<<grid, threads>>> (nThreads, nStreams,
		                                            d_in, d_out);
			break;
		case 72:
			/* intensity 18.0 */
			intensity_kernel_1_72 <<<grid, threads>>> (nThreads, nStreams,
		                                            d_in, d_out);
			break;
		case 128:
			/* intensity 32.0 */
			intensity_kernel_1_128 <<<grid, threads>>> (nThreads, nStreams,
		                                            d_in, d_out);
			break;
		case 160:
			/* intensity 40.0 */
			intensity_kernel_1_160 <<<grid, threads>>> (nThreads, nStreams,
		                                            d_in, d_out);
			break;

		case 256:
			/* intensity 64.0 */
			intensity_kernel_1_256 <<<grid, threads>>> (nThreads, nStreams,
		                                            d_in, d_out);
			break;
		case 512:
			/* intensity 128.0 */
			intensity_kernel_1_512 <<<grid, threads>>> (nThreads, nStreams,
		                                            d_in, d_out);
			break;
		case 1024:
			/* intensity 256.0 */
			intensity_kernel_1_1024 <<<grid, threads>>> (nThreads, nStreams,
		                                            d_in, d_out);
			break;
		case 2048:
			/* intensity 512.0 */
			intensity_kernel_1_2048 <<<grid, threads>>> (nThreads, nStreams,
		                                            d_in, d_out);
			break;
		case 4096:
			/* intensity 1024.0 */
			intensity_kernel_1_4096 <<<grid, threads>>> (nThreads, nStreams,
		                                            d_in, d_out);
			break;
		default:
			fprintf(stderr, "Invalid combination: %d nStreams and %d nFlops\n",
							nStreams, nFlops);
			break;
		}

  // end timer
  cutilSafeCall (hipEventRecord (stop, 0));
  /* cutilSafeCall (cutilDeviceSynchronize ()); */
  cutilSafeCall (hipDeviceSynchronize ());
  cutilSafeCall (hipEventElapsedTime (&total_time_taken, start, stop));

	return total_time_taken;
}

int main(int argc, char** argv)
{	
	int i;

	// timer
	float total_time_taken;

  // execution parameters
  double intensity;
  int nThreads;
  int bSize;
  int nStreams;
	int nFlops;

	int num_blocks;

	// memory
	TYPE* h_in;
	TYPE* h_out;
	TYPE* d_in;
	TYPE* d_out;

	// file
	FILE* fp;

	if(argc != 3) {
		fprintf(stderr, "usage: %s <file name> <intensity>\n", argv[0]);
		exit (0);
	}

	// intensity is flop-to-byte ratio
	intensity = atof (argv[2]);
	nFlops = (intensity * sizeof (TYPE)) / 4;

	// input file
	fp = fopen (argv[1], "r");

  if(fp == NULL) {
    fprintf(stderr, "File %s does not exist, exiting...\n", argv[1]);
		exit (0);
  } else {
    fprintf(stderr, "Reading parameters...\n");
    readConfig (fp, &nThreads, &bSize, &nStreams);
    fclose (fp);
  }

	fprintf(stderr,
					"Best bandwidth at %d threads %d block size %d streams/thread\n",
					nThreads, bSize, nStreams);
	fprintf(stderr, "Required flops for intensity of %.2f: %d\n", intensity, 
					nFlops);

	num_blocks = (nThreads * nStreams + bSize - 1) / bSize;
	fprintf(stderr, "Number of thread blocks: %d\n", num_blocks);

	// find the best GPU in the system
  fprintf(stderr, "Running on device: %d\n", cutGetMaxGflopsDeviceId ());
	hipSetDevice(cutGetMaxGflopsDeviceId ());


	// allocate memory
	h_in = (TYPE*) malloc (nThreads * nStreams * sizeof (TYPE));
	h_out = (TYPE*) malloc (nThreads * nStreams * sizeof (TYPE));
	cutilSafeCall (hipMalloc ((void**) &d_in, nThreads * nStreams *
														 sizeof (TYPE)));
	cutilSafeCall (hipMalloc ((void**) &d_out, nThreads * nStreams *
														 sizeof (TYPE)));

	// initialize memory
	for(i = 0; i < nThreads * nStreams; i++) {
    h_in[i] = make_float4 (drand48 (), drand48 (), drand48 (), drand48 ());
    h_out[i] = make_float4 (0.0f, 0.0f, 0.0f, 0.0f);
	}
	cutilSafeCall (hipMemcpy (d_in, h_in, nThreads * nStreams * sizeof (TYPE),
														 hipMemcpyHostToDevice));
	cutilSafeCall (hipMemcpy (d_out, h_out, nThreads * nStreams * sizeof (TYPE),
														 hipMemcpyHostToDevice));


	// Execute kernel
	total_time_taken = 0.0;
	total_time_taken = executeKernel (num_blocks, bSize, nThreads, nStreams, 
																			 nFlops, d_in, d_out);


  gpu_check_error__srcpos (stderr, __FILE__, __LINE__);

  // copy results back
  cutilSafeCall (hipMemcpy (h_out, d_out, nThreads * nStreams * sizeof (TYPE),
                             hipMemcpyDeviceToHost));

  printf("Time taken to trasnfer %f GB and execute %f Gflops: %f (ms)\n", 
				 (((2.0 * nThreads / 1e6) * nStreams * sizeof (TYPE) ) / 1e3), 
				 (((2.0 * nThreads / 1e6) * nStreams * nFlops * 4) / 1e3),
				 total_time_taken);
  printf("Effective performance: %f (GB/s)\n", (((2.0 * nThreads / 1e6) *
         nStreams * sizeof (TYPE) )) / total_time_taken);
  fprintf(stderr, "Effective performance: %f (GFlops/s)\n",
         (((2.0 * nThreads / 1e6) * nFlops * 4 * nStreams)) / total_time_taken);


  free (h_in);
  free (h_out);
  cutilSafeCall (hipFree (d_in));
  cutilSafeCall (hipFree (d_out));


	fprintf(stderr, "Target intensity is %.2f\n", intensity);
	return 0;
}
