#include "hip/hip_runtime.h"
/*
MIT License
Copyright 2020 Jee W. Choi, Marat Dukhan, and Xing Liu
Permission is hereby granted, free of charge, to any person obtaining a copy of 
this software and associated documentation files (the "Software"), to deal in 
the Software without restriction, including without limitation the rights to use, 
copy, modify, merge, publish, distribute, sublicense, and/or sell copies of the 
Software, and to permit persons to whom the Software is furnished to do so, subject 
to the following conditions:
The above copyright notice and this permission notice shall be included in all 
copies or substantial portions of the Software.
THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR IMPLIED, 
INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS FOR A 
PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT 
HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF 
CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE 
OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
*/

// libraries
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <cutil_inline.h>

#include "compute_kernel.h"

// GPU kernel error checking function
void gpu_check_error__srcpos (FILE* fp, const char* filename, size_t line)
{
  hipError_t C_E = hipGetLastError ();
  if (C_E) {
    fprintf (fp, "*** [%s:%lu] CUDA ERROR %d: %s ***\n", filename, line, C_E, 
						 hipGetErrorString (C_E));
    fflush (fp);
    exit (-1); /* abort program */
  }
}


float runTest(int num_threads, int block_size, int num_flops)
{
	int num_blocks = (num_threads + block_size - 1) / block_size;

	TYPE* h_out;
	TYPE* d_out;

	// timer
	hipEvent_t start, stop;
	float total_time_taken;

	if(num_blocks <= 65535) {

		// allocate memory
		h_out = (TYPE*) malloc (num_threads * sizeof (TYPE));
		cutilSafeCall (hipMalloc ((void**) &d_out, num_threads * sizeof (TYPE)));

		dim3 grid (num_blocks);
		dim3 threads (block_size);

		// start timer
    cutilSafeCall (hipEventCreate (&start));
    cutilSafeCall (hipEventCreate (&stop));
    cutilSafeCall (hipEventRecord (start, 0));

		// execute kernel
		switch (num_flops) {
			case 1:
				compute_kernel_1 <<<grid, threads>>> (num_threads, num_flops, d_out);
				break;
			case 2:
				compute_kernel_2 <<<grid, threads>>> (num_threads, num_flops, d_out);
				break;
			case 4:
				compute_kernel_4 <<<grid, threads>>> (num_threads, num_flops, d_out);
				break;
			case 8:
				compute_kernel_8 <<<grid, threads>>> (num_threads, num_flops, d_out);
				break;
			case 16:
				compute_kernel_16 <<<grid, threads>>> (num_threads, num_flops, d_out);
				break;
			case 32:
				compute_kernel_32 <<<grid, threads>>> (num_threads, num_flops, d_out);
				break;
			case 64:
				compute_kernel_64 <<<grid, threads>>> (num_threads, num_flops, d_out);
				break;
			case 128:
				compute_kernel_128 <<<grid, threads>>> (num_threads, num_flops, d_out);
				break;
			case 256:
				compute_kernel_256 <<<grid, threads>>> (num_threads, num_flops, d_out);
				break;
			case 512:
				compute_kernel_512 <<<grid, threads>>> (num_threads, num_flops, d_out);
				break;
			case 1024:
				compute_kernel_1024 <<<grid, threads>>> (num_threads, num_flops, d_out);
				break;
			case 2048:
				compute_kernel_2048 <<<grid, threads>>> (num_threads, num_flops, d_out);
				break;
			case 4096:
				compute_kernel_4096 <<<grid, threads>>> (num_threads, num_flops, d_out);
				break;
			default:
				fprintf(stderr, "Invalid num_flops: %d\n", num_flops);
				total_time_taken = -1.0f;
				break;
		}
		// end timer
    cutilSafeCall (hipEventRecord (stop, 0));
    // cutilSafeCall (cutilDeviceSynchronize ());
    cutilSafeCall (hipDeviceSynchronize ());
    cutilSafeCall (hipEventElapsedTime (&total_time_taken, start, stop));

		gpu_check_error__srcpos (stderr, __FILE__, __LINE__);

		free (h_out);
		cutilSafeCall (hipFree (d_out));

		fprintf(stderr, 
						"Execution configuration: %d blocks %d threads %d flops: %f (ms)\n",
						num_blocks, block_size, num_flops, total_time_taken);

	} else {
		total_time_taken = -1.0f;
	}

	return total_time_taken;
}



void autotune(int* num_threads, int* block_size, int* flops_per_thread)
{
	int i, j, k, l;

	// structures for storing execution configurations
	int *threads;
	int *tb;
	int *num_flops;
  float* total_time_taken;
	float* perf;
	float best_perf;

	// allocate memory for storing kernel execution configuration 
	threads = (int*) malloc (5 * 3 * 13 * sizeof (int));
	tb = (int*) malloc (5 * 3 * 13 * sizeof (int));
	num_flops = (int*) malloc (5 * 3 * 13 * sizeof (int));
	total_time_taken = (float*) malloc (5 * 3 * 13 * sizeof (float));
	perf = (float*) malloc (5 * 3 * 13 * sizeof (float));

	// autotune for best execution configuration
	l = 0;
	// vary # of threads
	for(i = 1048576; i <= 16777216; i *= 2) {
		// vary thread block size
		for(j = 128; j <= 512; j *= 2) {
			for(k = 1; k <= 4096; k *= 2) {
				threads[l] = i;
				tb[l] = j;
				num_flops[l] = k;

				// wrapper function for the test kernels
				total_time_taken[l] = runTest (i, j, k);
			
				if(total_time_taken[l] > 0) {
					perf[l] = ((((2.0 * i / 1e6) * k)) / total_time_taken[l]);
				} else {
					perf[l] = 0.0;
				}
				l++;	
			}
		}
	}	

	// find best performing configuration
	best_perf = perf[0];
	// j keeps track of the index to the best configuration
	j = 0;
	for(int i = 1; i < l; i++) {
		if(perf[i] > best_perf) {
			best_perf = perf[i];
			j = i;
		}
	}
	*num_threads = threads[j];
	*block_size = tb[j];
	*flops_per_thread = num_flops[j];

	free (threads);
	free (tb);
	free (num_flops);
	free (total_time_taken);
	free (perf);
}

void writeConfig(char* file_name, int num_threads, int block_size, 
								 int flops_per_thread) 
{
	FILE* fp = fopen(file_name, "w");
	fprintf(fp, "%d\n", num_threads);
	fprintf(fp, "%d\n", block_size);
	fprintf(fp, "%d\n", flops_per_thread);
	fclose (fp);
}

void readConfig(FILE *fp, int* num_threads, int* block_size, 
								int* flops_per_thread) 
{
	char line[20];
	if(fgets(line, 20, fp) != NULL) {
		*num_threads = atoi (line);
	} else {
		fprintf(stderr, "Can't find number of threads in the config file\n");
		exit(0);
	}
	if(fgets(line, 20, fp) != NULL) {
		*block_size = atoi (line);
	} else {
		fprintf(stderr, "Can't find block size in the config file\n");
		exit(0);
	}
	if(fgets(line, 20, fp) != NULL) {
		*flops_per_thread = atoi (line);
	} else {
		fprintf(stderr, "Can't find flops per thread in the config file\n");
		exit(0);
	}
}

#if 0
int validateResults(TYPE* out, int nThreads, int bSize, int nFlops)
{
	int i, j, k;
	int index;
	TYPE tmp1, tmp2;

	fprintf(stderr, "generating random numbers from 0 to %d\n", RAND_MAX);

	// test 1000 samples
	k = 0;
	for(i = 0; i < 1000; i++) {
		// index = rand () % nThreads;	
		index = i;
		tmp1 = 1.0 * (index % bSize);
		tmp2 = 2.0 * (index % bSize);
		for(j = 0; j < (nFlops / 2); j++) {
			tmp1 = tmp1 + tmp1 * CONST;
			tmp2 = tmp2 + tmp2 * CONST;
		}
		if(abs ((tmp1 + tmp2) - out[index]) > 1e-5) {
			k++;
			// fprintf(stderr, "%d ==> %f %f\n", index, (tmp1 + tmp2), out[index]);
		} else {
			// fprintf(stderr, "%d <== %f %f\n", index, (tmp1 + tmp2), out[index]);
		}
	}
	return k;
}
#endif
int main(int argc, char** argv)
{
	int i;

	// timer
	hipEvent_t start, stop;
	float total_time_taken;

	// execution parameters
	int nThreads;
	int bSize;
	int nFlops;

	int num_blocks;

	// memory data structures
	TYPE* h_in;
	TYPE* d_in;
  TYPE* d_out;
  TYPE* h_out;

	// file
	FILE* fp;

  if(argc != 2) {
    fprintf(stderr, "usage: %s <parameter list file>\n", argv[0]);
    exit (0);
  }

	fp = fopen (argv[1], "r");

	if(fp == NULL) {
		fprintf(stderr, "File %s does not exist, autotuning...\n", argv[1]);	
		autotune (&nThreads, &bSize, &nFlops);
		writeConfig (argv[1], nThreads, bSize, nFlops);
	} else {
		fprintf(stderr, "Reading parameters...\n");
		readConfig (fp, &nThreads, &bSize, &nFlops);
		fclose (fp);
	}

	fprintf(stderr, 
					"Best performance at %d threads %d block size %d flops/thread\n",
					nThreads, bSize, nFlops);
	
	// find the best GPU in the system
	hipSetDevice(cutGetMaxGflopsDeviceId ());

	num_blocks = (nThreads + bSize - 1) / bSize;

	// allocate memory
	h_in = (TYPE*) malloc (nThreads * sizeof (TYPE));
	h_out = (TYPE*) malloc (nThreads * sizeof (TYPE));
	cutilSafeCall (hipMalloc ((void**) &d_in, nThreads * sizeof (TYPE)));
	cutilSafeCall (hipMalloc ((void**) &d_out, nThreads * sizeof (TYPE)));

	// initialize memory
	for(i = 0; i < nThreads; i++) {
		h_in[i] = drand48 ();
	}
	cutilSafeCall (hipMemcpy (d_in, h_in, nThreads * sizeof (TYPE), 
														 hipMemcpyHostToDevice));

	
  dim3 grid (num_blocks);
  dim3 threads (bSize);

  // start timer
	cutilSafeCall (hipEventCreate (&start));
	cutilSafeCall (hipEventCreate (&stop));
	cutilSafeCall (hipEventRecord (start, 0));

		for(int iter = 0; iter < NUM_ITER; iter++) {
		switch (nFlops) {
			case 1:
				compute_kernel_1 <<<grid, threads>>> (nThreads, nFlops, d_out);
				break;
			case 2:
				compute_kernel_2 <<<grid, threads>>> (nThreads, nFlops, d_out);
				break;
			case 4:
				compute_kernel_4 <<<grid, threads>>> (nThreads, nFlops, d_out);
				break;
			case 8:
				compute_kernel_8 <<<grid, threads>>> (nThreads, nFlops, d_out);
				break;
			case 16:
				compute_kernel_16 <<<grid, threads>>> (nThreads, nFlops, d_out);
				break;
			case 32:
				compute_kernel_32 <<<grid, threads>>> (nThreads, nFlops, d_out);
				break;
			case 64:
				compute_kernel_64 <<<grid, threads>>> (nThreads, nFlops, d_out);
				break;
			case 128:
				compute_kernel_128 <<<grid, threads>>> (nThreads, nFlops, d_out);
				break;
			case 256:
				compute_kernel_256 <<<grid, threads>>> (nThreads, nFlops, d_out);
				break;
			case 512:
				compute_kernel_512 <<<grid, threads>>> (nThreads, nFlops, d_out);
				break;
			case 1024:
				compute_kernel_1024 <<<grid, threads>>> (nThreads, nFlops, d_out);
				break;
			case 2048:
				compute_kernel_2048 <<<grid, threads>>> (nThreads, nFlops, d_out);
				break;
			case 4096:
				compute_kernel_4096 <<<grid, threads>>> (nThreads, nFlops, d_out);
				break;
			default:
				fprintf(stderr, "Invalid nFlops: %d\n", nFlops);
				total_time_taken = -1.0f;
				break;
		}
	}

  // end timer
  cutilSafeCall (hipEventRecord (stop, 0));
  // cutilSafeCall (cutilDeviceSynchronize ());
  cutilSafeCall (hipDeviceSynchronize ());
  cutilSafeCall (hipEventElapsedTime (&total_time_taken, start, stop));
  cutilSafeCall (hipDeviceSynchronize ());

	total_time_taken = total_time_taken / NUM_ITER;

  gpu_check_error__srcpos (stderr, __FILE__, __LINE__);

  // copy results back
  cutilSafeCall (hipMemcpy (h_out, d_out, nThreads * sizeof (TYPE),
                             hipMemcpyDeviceToHost));

	/*
	fprintf(stderr, "Results validated: %d\n", validateResults (h_out, nThreads, 
																															bSize, nFlops));
	 */

  fprintf(stderr, "Time taken to execute %f gflops: %f (ms)\n", (((2.0 * 
					nThreads / 1e6) * nFlops) / 1e3), total_time_taken);
  fprintf(stderr, "Effective performance: %f (GFlops/s)\n",
         (((2.0 * nThreads / 1e6) * nFlops)) / total_time_taken);

	free (h_out);
	cutilSafeCall (hipFree (d_out));

	return 0;
}
