#include "hip/hip_runtime.h"
// libraries
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <cutil_inline.h>

#include "bandwidthilp_kernel.h"

// GPU kernel error checking function
void gpu_check_error__srcpos (FILE* fp, const char* filename, size_t line)
{
  hipError_t C_E = hipGetLastError ();
  if (C_E) {
    fprintf (fp, "*** [%s:%lu] CUDA ERROR %d: %s ***\n", filename, line, C_E, 
						 hipGetErrorString (C_E));
    fflush (fp);
    exit (-1); /* abort program */
  }
}

float runTest(int num_threads, int block_size, int num_streams)
{
	int i;

	int num_blocks = (num_threads + block_size - 1) / block_size;

	TYPE* h_in;
	TYPE* h_out;
	TYPE* d_in;
	TYPE* d_out;

	// timer
	hipEvent_t start, stop;
	float total_time_taken;

	if((num_blocks <= 2147483647) && 
		 (((num_threads / 1024) * num_streams * sizeof (TYPE)) < (7e5))) {
		fprintf(stderr, "Execution configuration: %d blocks %d threads %d streams\t"
						, num_blocks, block_size, num_streams);


  	hipSetDevice(cutGetMaxGflopsDeviceId ());

		// allocate memory
		h_in = (TYPE*) malloc (num_threads * num_streams * sizeof (TYPE));
		h_out = (TYPE*) malloc (num_threads * num_streams * sizeof (TYPE));
		cutilSafeCall (hipMalloc ((void**) &d_in, num_threads * num_streams * 
															 sizeof (TYPE)));
		cutilSafeCall (hipMalloc ((void**) &d_out, num_threads * num_streams * 
															 sizeof (TYPE)));

		// initialize memory
		for(i = 0; i < num_threads * num_streams; i++) {
			h_in[i] = make_float4 (1.0f, 1.0f, 1.0f, 1.0f);
			h_out[i] = make_float4 (0.0f, 0.0f, 0.0f, 0.0f);
			/*
			h_in[i] = 1.0;
			h_out[i] = 0.0;
			 */
		}
		cutilSafeCall (hipMemcpy (d_in, h_in, num_threads * num_streams * 
									 						 sizeof (TYPE), hipMemcpyHostToDevice));
		cutilSafeCall (hipMemcpy (d_out, h_out, num_threads * num_streams * 
															 sizeof (TYPE), hipMemcpyHostToDevice));

		dim3 grid (num_blocks);
		dim3 threads (block_size);

		// start timer
		cutilSafeCall (hipEventCreate (&start));
		cutilSafeCall (hipEventCreate (&stop));
		cutilSafeCall (hipEventRecord (start, 0));

		// execute kernel
		switch (num_streams) {
			case 1:
				bandwidth_kernel_1 <<<grid, threads>>> (num_threads, num_streams,
																								d_in, d_out);
				break;
			case 2:
				bandwidth_kernel_2 <<<grid, threads>>> (num_threads, num_streams,
																								d_in, d_out);
				break;
			case 4:
				bandwidth_kernel_4 <<<grid, threads>>> (num_threads, num_streams,
																								d_in, d_out);
				break;
			case 8:
				bandwidth_kernel_8 <<<grid, threads>>> (num_threads, num_streams,
																								d_in, d_out);
				break;
			case 16:
				bandwidth_kernel_16 <<<grid, threads>>> (num_threads, num_streams,
																								d_in, d_out);
				break;
			case 32:
				bandwidth_kernel_32 <<<grid, threads>>> (num_threads, num_streams,
																								d_in, d_out);
				break;
			case 64:
				bandwidth_kernel_64 <<<grid, threads>>> (num_threads, num_streams,
																								d_in, d_out);
				break;
			case 128:
				bandwidth_kernel_128 <<<grid, threads>>> (num_threads, num_streams,
																									d_in, d_out);
				break;
			default:
				fprintf(stderr, "Invalid num_streams: %d\n", num_streams);
				total_time_taken = 1.0f;
				break;
		}


		// end timer
		cutilSafeCall (hipEventRecord (stop, 0));
		// cutilSafeCall (cutilDeviceSynchronize ());
		cutilSafeCall (hipDeviceSynchronize ());
		cutilSafeCall (hipEventElapsedTime (&total_time_taken, start, stop));

		gpu_check_error__srcpos (stderr, __FILE__, __LINE__);

		cutilSafeCall (hipMemcpy (h_out, d_out, num_threads * num_streams * 
															 sizeof (TYPE), hipMemcpyDeviceToHost));

		free (h_in);
		free (h_out);
		cutilSafeCall (hipFree (d_in));
		cutilSafeCall (hipFree (d_out));

		fprintf(stderr, "...Done\n");
	} else {
		total_time_taken = -1.0f;
	}

	return total_time_taken;
}



void autotune(int* num_threads, int* block_size, int* streams_per_thread)
{
	int i, j, k, l;

	// structures for storing execution configurations
	int *threads;
	int *tb;
	int *num_streams;
  float* total_time_taken;
	float* perf;
	float best_perf;


	// allocate memory for storing kernel execution configuration 
	threads = (int*) malloc (5 * 3 * 9 * sizeof (int));
	tb = (int*) malloc (5 * 3 * 9 * sizeof (int));
	num_streams = (int*) malloc (5 * 3 * 9 * sizeof (int));
	total_time_taken = (float*) malloc (5 * 3 * 9 * sizeof (float));
	perf = (float*) malloc (5 * 3 * 9 * sizeof (float));

	// auto-tune for best execution configuration
	l = 0;
	// vary # of threads
	for(i = 1048576; i <= 33554432; i *= 2) {
		// vary thread block size
		for(j = 128; j <= 1024; j *= 2) {
			// vary streams per thread
			for(k = 1; k <= 8; k *= 2) {
				threads[l] = i;
				tb[l] = j;
				num_streams[l] = k;

				// wrapper function for the test kernels
				total_time_taken[l] = runTest (i, j, k);
			
				if(total_time_taken[l] > 0) {
					perf[l] = ((((2.0 * i * sizeof (TYPE) / 1e6) * k)) / 
										 total_time_taken[l]);
				} else {
					perf[l] = 0.0;
				}
				l++;	
			}
		}
	}	

	// find best performing configuration
	best_perf = perf[0];
	// j keeps track of the index to the best configuration
	j = 0;
	for(int i = 1; i < l; i++) {
		if(perf[i] > best_perf) {
			best_perf = perf[i];
			j = i;
		}
	}
	*num_threads = threads[j];
	*block_size = tb[j];
	*streams_per_thread = num_streams[j];

	free (threads);
	free (tb);
	free (num_streams);
	free (total_time_taken);
	free (perf);
}

void writeConfig(char* file_name, int num_threads, int block_size, 
								 int streams_per_thread) 
{
	FILE* fp = fopen(file_name, "w");
	fprintf(fp, "%d\n", num_threads);
	fprintf(fp, "%d\n", block_size);
	fprintf(fp, "%d\n", streams_per_thread);
	fclose (fp);
}

void readConfig(FILE *fp, int* num_threads, int* block_size, 
								int* streams_per_thread) 
{
	char line[20];
	if(fgets(line, 20, fp) != NULL) {
		*num_threads = atoi (line);
	} else {
		fprintf(stderr, "Can't find number of threads in the config file\n");
		exit(0);
	}
	if(fgets(line, 20, fp) != NULL) {
		*block_size = atoi (line);
	} else {
		fprintf(stderr, "Can't find block size in the config file\n");
		exit(0);
	}
	if(fgets(line, 20, fp) != NULL) {
		*streams_per_thread = atoi (line);
	} else {
		fprintf(stderr, "Can't find streams per thread in the config file\n");
		exit(0);
	}
}

int validateResults (TYPE* in, TYPE* out, int n)
{
	int i, j;
	j = 0;
	for(i = 0; i < n; i++) {
		// if(in[i] != out[i]) j++;
	}
	return j;
}

int main(int argc, char** argv)
{
	int i;

	// timer
	float total_time_taken;
	hipEvent_t start, stop;

	// execution parameters
	int nThreads;
	int bSize;
	int nStreams;

	int num_blocks;

	// memory data structures
	TYPE* d_in;
	TYPE* h_in;
  TYPE* d_out;
  TYPE* h_out;

	// file
	FILE* fp;

  if(argc != 2) {
    fprintf(stderr, "usage: %s <parameter list file>\n", argv[0]);
    exit (0);
  }

	fp = fopen (argv[1], "r");

	if(fp == NULL) {
		fprintf(stderr, "File %s does not exist, autotuning...\n", argv[1]);	
		autotune (&nThreads, &bSize, &nStreams);
		writeConfig (argv[1], nThreads, bSize, nStreams);
	} else {
		fprintf(stderr, "Reading parameters...\n");
		readConfig (fp, &nThreads, &bSize, &nStreams);
		fclose (fp);
	}

	fprintf(stderr, 
					"Best performance at %d threads %d block size %d streams/thread\n",
					nThreads, bSize, nStreams);

	// find the best GPU in the system
  hipSetDevice(cutGetMaxGflopsDeviceId ());

	num_blocks = (nThreads + bSize - 1) / bSize;

	// allocate memory
	h_in = (TYPE*) malloc (nThreads * nStreams * sizeof (TYPE));
	h_out = (TYPE*) malloc (nThreads * nStreams * sizeof (TYPE));
	cutilSafeCall (hipMalloc ((void**) &d_in, nThreads * nStreams * 
														 sizeof (TYPE)));
	cutilSafeCall (hipMalloc ((void**) &d_out, nThreads * nStreams * 
														 sizeof (TYPE)));

	// initialize memory
	for(i = 0; i < nThreads * nStreams; i++) {
		h_in[i] = make_float4 (drand48 (), drand48 (), drand48 (), drand48 ());
		h_out[i] = make_float4 (0.0f, 0.0f, 0.0f, 0.0f);
	}
	cutilSafeCall (hipMemcpy (d_in, h_in, nThreads * nStreams * sizeof (TYPE),
														 hipMemcpyHostToDevice));
	cutilSafeCall (hipMemcpy (d_out, h_out, nThreads * nStreams * sizeof (TYPE),
														 hipMemcpyHostToDevice));


  dim3 grid (num_blocks);
  dim3 threads (bSize);


  // start timer
	cutilSafeCall (hipEventCreate (&start));
	cutilSafeCall (hipEventCreate (&stop));
	cutilSafeCall (hipEventRecord (start, 0));

	for(int cnt = 0; cnt < NUM_ITER; cnt++) 
	switch (nStreams) {
		case 1:
			bandwidth_kernel_1 <<<grid, threads>>> (nThreads, nStreams,
																							d_in, d_out);
			break;
		case 2:
			bandwidth_kernel_2 <<<grid, threads>>> (nThreads, nStreams,
																							d_in, d_out);
			break;
		case 4:
			bandwidth_kernel_4 <<<grid, threads>>> (nThreads, nStreams,
																							d_in, d_out);
			break;
		case 8:
			bandwidth_kernel_8 <<<grid, threads>>> (nThreads, nStreams,
																							d_in, d_out);
				break;
		case 16:
			bandwidth_kernel_16 <<<grid, threads>>> (nThreads, nStreams,
																							 d_in, d_out);
			break;
		case 32:
			bandwidth_kernel_32 <<<grid, threads>>> (nThreads, nStreams,
																							 d_in, d_out);
			break;
		case 64:
			bandwidth_kernel_64 <<<grid, threads>>> (nThreads, nStreams,
																							 d_in, d_out);
			break;
		case 128:
			bandwidth_kernel_128 <<<grid, threads>>> (nThreads, nStreams,
																								d_in, d_out);
			break;
		default:
			fprintf(stderr, "Invalid nStreams: %d\n", nStreams);
			total_time_taken = -1.0f;
			break;
		}
	
  // end timer
	cutilSafeCall (hipEventRecord (stop, 0));
	cutilSafeCall (hipDeviceSynchronize ());
	cutilSafeCall (hipEventElapsedTime (&total_time_taken, start, stop));
	total_time_taken = total_time_taken / NUM_ITER;

	// Check for kernel errors
  gpu_check_error__srcpos (stderr, __FILE__, __LINE__);

  // copy results back
  cutilSafeCall (hipMemcpy (h_out, d_out, nThreads * nStreams * sizeof (TYPE),
                             hipMemcpyDeviceToHost));

	printf("Results validated: %d\n", validateResults (h_in, h_out, 
																										 nThreads * nStreams));

  printf("Time taken to trasnfer %f GB: %f (ms)\n", (((2.0 * nThreads /
         1e6) * nStreams * sizeof (TYPE) ) / 1e3), total_time_taken);
  printf("Effective performance: %f (GB/s)\n", (((2.0 * nThreads / 1e6) * 
				 nStreams * sizeof (TYPE) )) / total_time_taken);

	free (h_in);
	free (h_out);
	cutilSafeCall (hipFree (d_in));
	cutilSafeCall (hipFree (d_out));


	return 0;
}
