#include "hip/hip_runtime.h"
/*
MIT License
Copyright 2020 Jee W. Choi, Marat Dukhan, and Xing Liu
Permission is hereby granted, free of charge, to any person obtaining a copy of 
this software and associated documentation files (the "Software"), to deal in 
the Software without restriction, including without limitation the rights to use, 
copy, modify, merge, publish, distribute, sublicense, and/or sell copies of the 
Software, and to permit persons to whom the Software is furnished to do so, subject 
to the following conditions:
The above copyright notice and this permission notice shall be included in all 
copies or substantial portions of the Software.
THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR IMPLIED, 
INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS FOR A 
PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT 
HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF 
CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE 
OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
*/

// #include "bandwidthilp_kernel.h"

__global__ void bandwidth_kernel_1 (int num_threads, int num_streams, TYPE* in,
																	TYPE* out)
{
	uint tid = threadIdx.x + blockIdx.x * blockDim.x;
	if(tid < num_threads) {
		out[tid] = in[tid];
		// memcpy (&out[tid], &in[tid], sizeof (TYPE));
	}
}

__global__ void bandwidth_kernel_2 (int num_threads, int num_streams, TYPE* in,
																	TYPE* out)
{
	uint tid = threadIdx.x + blockIdx.x * blockDim.x;
	uint offset = blockIdx.x * blockDim.x * 2;

	if(tid < num_threads) {
		out[offset + threadIdx.x] = in[offset + threadIdx.x];
		out[offset + blockDim.x + threadIdx.x] = in[offset + blockDim.x + 
																								threadIdx.x];
		// out[num_threads + tid] = in[num_threads + tid];
	}
}

__global__ void bandwidth_kernel_4 (int num_threads, int num_streams, TYPE* in,
																	TYPE* out)
{
	uint tid = threadIdx.x + blockIdx.x * blockDim.x;
	uint offset = blockIdx.x * blockDim.x * 4;

	if(tid < num_threads) {
		out[offset + threadIdx.x] = in[offset + threadIdx.x];
		out[offset + blockDim.x + threadIdx.x] = in[offset + blockDim.x + 
																								threadIdx.x];
		out[offset + 2 * blockDim.x + threadIdx.x] = in[offset + 2 * blockDim.x + 
																										threadIdx.x];
		out[offset + 3 * blockDim.x + threadIdx.x] = in[offset + 3 * blockDim.x + 
																										threadIdx.x];

		/*
		out[tid] = in[tid];
		out[num_threads + tid] = in[num_threads + tid];
		out[2 * num_threads + tid] = in[2 * num_threads + tid];
		out[3 * num_threads + tid] = in[3 * num_threads + tid];
		 */
	}
}

__global__ void bandwidth_kernel_8 (int num_threads, int num_streams, TYPE* in,
																	TYPE* out)
{
	uint tid = threadIdx.x + blockIdx.x * blockDim.x;
	uint offset = blockIdx.x * blockDim.x * 8;

	if(tid < num_threads) {
		out[offset + threadIdx.x] = in[offset + threadIdx.x];
		out[offset + blockDim.x + threadIdx.x] = in[offset + blockDim.x + 
																								threadIdx.x];
		out[offset + 2 * blockDim.x + threadIdx.x] = in[offset + 2 * blockDim.x + 
																										threadIdx.x];
		out[offset + 3 * blockDim.x + threadIdx.x] = in[offset + 3 * blockDim.x + 
																										threadIdx.x];
		out[offset + 4 * blockDim.x + threadIdx.x] = in[offset + 4 * blockDim.x + 
																										threadIdx.x];
		out[offset + 5 * blockDim.x + threadIdx.x] = in[offset + 5 * blockDim.x + 
																										threadIdx.x];
		out[offset + 6 * blockDim.x + threadIdx.x] = in[offset + 6 * blockDim.x + 
																										threadIdx.x];
		out[offset + 7 * blockDim.x + threadIdx.x] = in[offset + 7 * blockDim.x + 
																										threadIdx.x];

		/*
		out[tid] = in[tid];
		out[num_threads + tid] = in[num_threads + tid];
		out[2 * num_threads + tid] = in[2 * num_threads + tid];
		out[3 * num_threads + tid] = in[3 * num_threads + tid];
		out[4 * num_threads + tid] = in[4 * num_threads + tid];
		out[5 * num_threads + tid] = in[5 * num_threads + tid];
		out[6 * num_threads + tid] = in[6 * num_threads + tid];
		out[7 * num_threads + tid] = in[7 * num_threads + tid];
		 */
	}
}

__global__ void bandwidth_kernel_16 (int num_threads, int num_streams, TYPE* in,
																	TYPE* out)
{
	uint tid = threadIdx.x + blockIdx.x * blockDim.x;

	if(tid < num_threads) {
		out[tid] = in[tid];
		out[num_threads + tid] = in[num_threads + tid];
		out[2 * num_threads + tid] = in[2 * num_threads + tid];
		out[3 * num_threads + tid] = in[3 * num_threads + tid];
		out[4 * num_threads + tid] = in[4 * num_threads + tid];
		out[5 * num_threads + tid] = in[5 * num_threads + tid];
		out[6 * num_threads + tid] = in[6 * num_threads + tid];
		out[7 * num_threads + tid] = in[7 * num_threads + tid];
		out[8 * num_threads + tid] = in[8 * num_threads + tid];
		out[9 * num_threads + tid] = in[9 * num_threads + tid];
		out[10 * num_threads + tid] = in[10 * num_threads + tid];
		out[11 * num_threads + tid] = in[11 * num_threads + tid];
		out[12 * num_threads + tid] = in[12 * num_threads + tid];
		out[13 * num_threads + tid] = in[13 * num_threads + tid];
		out[14 * num_threads + tid] = in[14 * num_threads + tid];
		out[15 * num_threads + tid] = in[15 * num_threads + tid];
	}
}

__global__ void bandwidth_kernel_32 (int num_threads, int num_streams, TYPE* in,
																	TYPE* out)
{
	uint tid = threadIdx.x + blockIdx.x * blockDim.x;

	if(tid < num_threads) {
		out[tid] = in[tid];
		out[num_threads + tid] = in[num_threads + tid];
		out[2 * num_threads + tid] = in[2 * num_threads + tid];
		out[3 * num_threads + tid] = in[3 * num_threads + tid];
		out[4 * num_threads + tid] = in[4 * num_threads + tid];
		out[5 * num_threads + tid] = in[5 * num_threads + tid];
		out[6 * num_threads + tid] = in[6 * num_threads + tid];
		out[7 * num_threads + tid] = in[7 * num_threads + tid];
		out[8 * num_threads + tid] = in[8 * num_threads + tid];
		out[9 * num_threads + tid] = in[9 * num_threads + tid];
		out[10 * num_threads + tid] = in[10 * num_threads + tid];
		out[11 * num_threads + tid] = in[11 * num_threads + tid];
		out[12 * num_threads + tid] = in[12 * num_threads + tid];
		out[13 * num_threads + tid] = in[13 * num_threads + tid];
		out[14 * num_threads + tid] = in[14 * num_threads + tid];
		out[15 * num_threads + tid] = in[15 * num_threads + tid];
		out[16 * num_threads + tid] = in[16 * num_threads + tid];
		out[17 * num_threads + tid] = in[17 * num_threads + tid];
		out[18 * num_threads + tid] = in[18 * num_threads + tid];
		out[19 * num_threads + tid] = in[19 * num_threads + tid];
		out[20 * num_threads + tid] = in[20 * num_threads + tid];
		out[21 * num_threads + tid] = in[21 * num_threads + tid];
		out[22 * num_threads + tid] = in[22 * num_threads + tid];
		out[23 * num_threads + tid] = in[23 * num_threads + tid];
		out[24 * num_threads + tid] = in[24 * num_threads + tid];
		out[25 * num_threads + tid] = in[25 * num_threads + tid];
		out[26 * num_threads + tid] = in[26 * num_threads + tid];
		out[27 * num_threads + tid] = in[27 * num_threads + tid];
		out[28 * num_threads + tid] = in[28 * num_threads + tid];
		out[29 * num_threads + tid] = in[29 * num_threads + tid];
		out[30 * num_threads + tid] = in[30 * num_threads + tid];
		out[31 * num_threads + tid] = in[31 * num_threads + tid];
	}
}

__global__ void bandwidth_kernel_64 (int num_threads, int num_streams, TYPE* in,
																	TYPE* out)
{
	uint tid = threadIdx.x + blockIdx.x * blockDim.x;

	if(tid < num_threads) {
		out[tid] = in[tid];
		out[num_threads + tid] = in[num_threads + tid];
		out[2 * num_threads + tid] = in[2 * num_threads + tid];
		out[3 * num_threads + tid] = in[3 * num_threads + tid];
		out[4 * num_threads + tid] = in[4 * num_threads + tid];
		out[5 * num_threads + tid] = in[5 * num_threads + tid];
		out[6 * num_threads + tid] = in[6 * num_threads + tid];
		out[7 * num_threads + tid] = in[7 * num_threads + tid];
		out[8 * num_threads + tid] = in[8 * num_threads + tid];
		out[9 * num_threads + tid] = in[9 * num_threads + tid];
		out[10 * num_threads + tid] = in[10 * num_threads + tid];
		out[11 * num_threads + tid] = in[11 * num_threads + tid];
		out[12 * num_threads + tid] = in[12 * num_threads + tid];
		out[13 * num_threads + tid] = in[13 * num_threads + tid];
		out[14 * num_threads + tid] = in[14 * num_threads + tid];
		out[15 * num_threads + tid] = in[15 * num_threads + tid];
		out[16 * num_threads + tid] = in[16 * num_threads + tid];
		out[17 * num_threads + tid] = in[17 * num_threads + tid];
		out[18 * num_threads + tid] = in[18 * num_threads + tid];
		out[19 * num_threads + tid] = in[19 * num_threads + tid];
		out[20 * num_threads + tid] = in[20 * num_threads + tid];
		out[21 * num_threads + tid] = in[21 * num_threads + tid];
		out[22 * num_threads + tid] = in[22 * num_threads + tid];
		out[23 * num_threads + tid] = in[23 * num_threads + tid];
		out[24 * num_threads + tid] = in[24 * num_threads + tid];
		out[25 * num_threads + tid] = in[25 * num_threads + tid];
		out[26 * num_threads + tid] = in[26 * num_threads + tid];
		out[27 * num_threads + tid] = in[27 * num_threads + tid];
		out[28 * num_threads + tid] = in[28 * num_threads + tid];
		out[29 * num_threads + tid] = in[29 * num_threads + tid];
		out[30 * num_threads + tid] = in[30 * num_threads + tid];
		out[31 * num_threads + tid] = in[31 * num_threads + tid];
		out[32 * num_threads + tid] = in[32 * num_threads + tid];
		out[33 * num_threads + tid] = in[33 * num_threads + tid];
		out[34 * num_threads + tid] = in[34 * num_threads + tid];
		out[35 * num_threads + tid] = in[35 * num_threads + tid];
		out[36 * num_threads + tid] = in[36 * num_threads + tid];
		out[37 * num_threads + tid] = in[37 * num_threads + tid];
		out[38 * num_threads + tid] = in[38 * num_threads + tid];
		out[39 * num_threads + tid] = in[39 * num_threads + tid];
		out[40 * num_threads + tid] = in[40 * num_threads + tid];
		out[41 * num_threads + tid] = in[41 * num_threads + tid];
		out[42 * num_threads + tid] = in[42 * num_threads + tid];
		out[43 * num_threads + tid] = in[43 * num_threads + tid];
		out[44 * num_threads + tid] = in[44 * num_threads + tid];
		out[45 * num_threads + tid] = in[45 * num_threads + tid];
		out[46 * num_threads + tid] = in[46 * num_threads + tid];
		out[47 * num_threads + tid] = in[47 * num_threads + tid];
		out[48 * num_threads + tid] = in[48 * num_threads + tid];
		out[49 * num_threads + tid] = in[49 * num_threads + tid];
		out[50 * num_threads + tid] = in[50 * num_threads + tid];
		out[51 * num_threads + tid] = in[51 * num_threads + tid];
		out[52 * num_threads + tid] = in[52 * num_threads + tid];
		out[53 * num_threads + tid] = in[53 * num_threads + tid];
		out[54 * num_threads + tid] = in[54 * num_threads + tid];
		out[55 * num_threads + tid] = in[55 * num_threads + tid];
		out[56 * num_threads + tid] = in[56 * num_threads + tid];
		out[57 * num_threads + tid] = in[57 * num_threads + tid];
		out[58 * num_threads + tid] = in[58 * num_threads + tid];
		out[59 * num_threads + tid] = in[59 * num_threads + tid];
		out[60 * num_threads + tid] = in[60 * num_threads + tid];
		out[61 * num_threads + tid] = in[61 * num_threads + tid];
		out[62 * num_threads + tid] = in[62 * num_threads + tid];
		out[63 * num_threads + tid] = in[63 * num_threads + tid];
	}
}

__global__ void bandwidth_kernel_128 (int num_threads, int num_streams, TYPE* in,
																	TYPE* out)
{
	uint tid = threadIdx.x + blockIdx.x * blockDim.x;

	if(tid < num_threads) {
		out[tid] = in[tid];
		out[num_threads + tid] = in[num_threads + tid];
		out[2 * num_threads + tid] = in[2 * num_threads + tid];
		out[3 * num_threads + tid] = in[3 * num_threads + tid];
		out[4 * num_threads + tid] = in[4 * num_threads + tid];
		out[5 * num_threads + tid] = in[5 * num_threads + tid];
		out[6 * num_threads + tid] = in[6 * num_threads + tid];
		out[7 * num_threads + tid] = in[7 * num_threads + tid];
		out[8 * num_threads + tid] = in[8 * num_threads + tid];
		out[9 * num_threads + tid] = in[9 * num_threads + tid];
		out[10 * num_threads + tid] = in[10 * num_threads + tid];
		out[11 * num_threads + tid] = in[11 * num_threads + tid];
		out[12 * num_threads + tid] = in[12 * num_threads + tid];
		out[13 * num_threads + tid] = in[13 * num_threads + tid];
		out[14 * num_threads + tid] = in[14 * num_threads + tid];
		out[15 * num_threads + tid] = in[15 * num_threads + tid];
		out[16 * num_threads + tid] = in[16 * num_threads + tid];
		out[17 * num_threads + tid] = in[17 * num_threads + tid];
		out[18 * num_threads + tid] = in[18 * num_threads + tid];
		out[19 * num_threads + tid] = in[19 * num_threads + tid];
		out[20 * num_threads + tid] = in[20 * num_threads + tid];
		out[21 * num_threads + tid] = in[21 * num_threads + tid];
		out[22 * num_threads + tid] = in[22 * num_threads + tid];
		out[23 * num_threads + tid] = in[23 * num_threads + tid];
		out[24 * num_threads + tid] = in[24 * num_threads + tid];
		out[25 * num_threads + tid] = in[25 * num_threads + tid];
		out[26 * num_threads + tid] = in[26 * num_threads + tid];
		out[27 * num_threads + tid] = in[27 * num_threads + tid];
		out[28 * num_threads + tid] = in[28 * num_threads + tid];
		out[29 * num_threads + tid] = in[29 * num_threads + tid];
		out[30 * num_threads + tid] = in[30 * num_threads + tid];
		out[31 * num_threads + tid] = in[31 * num_threads + tid];
		out[32 * num_threads + tid] = in[32 * num_threads + tid];
		out[33 * num_threads + tid] = in[33 * num_threads + tid];
		out[34 * num_threads + tid] = in[34 * num_threads + tid];
		out[35 * num_threads + tid] = in[35 * num_threads + tid];
		out[36 * num_threads + tid] = in[36 * num_threads + tid];
		out[37 * num_threads + tid] = in[37 * num_threads + tid];
		out[38 * num_threads + tid] = in[38 * num_threads + tid];
		out[39 * num_threads + tid] = in[39 * num_threads + tid];
		out[40 * num_threads + tid] = in[40 * num_threads + tid];
		out[41 * num_threads + tid] = in[41 * num_threads + tid];
		out[42 * num_threads + tid] = in[42 * num_threads + tid];
		out[43 * num_threads + tid] = in[43 * num_threads + tid];
		out[44 * num_threads + tid] = in[44 * num_threads + tid];
		out[45 * num_threads + tid] = in[45 * num_threads + tid];
		out[46 * num_threads + tid] = in[46 * num_threads + tid];
		out[47 * num_threads + tid] = in[47 * num_threads + tid];
		out[48 * num_threads + tid] = in[48 * num_threads + tid];
		out[49 * num_threads + tid] = in[49 * num_threads + tid];
		out[50 * num_threads + tid] = in[50 * num_threads + tid];
		out[51 * num_threads + tid] = in[51 * num_threads + tid];
		out[52 * num_threads + tid] = in[52 * num_threads + tid];
		out[53 * num_threads + tid] = in[53 * num_threads + tid];
		out[54 * num_threads + tid] = in[54 * num_threads + tid];
		out[55 * num_threads + tid] = in[55 * num_threads + tid];
		out[56 * num_threads + tid] = in[56 * num_threads + tid];
		out[57 * num_threads + tid] = in[57 * num_threads + tid];
		out[58 * num_threads + tid] = in[58 * num_threads + tid];
		out[59 * num_threads + tid] = in[59 * num_threads + tid];
		out[60 * num_threads + tid] = in[60 * num_threads + tid];
		out[61 * num_threads + tid] = in[61 * num_threads + tid];
		out[62 * num_threads + tid] = in[62 * num_threads + tid];
		out[63 * num_threads + tid] = in[63 * num_threads + tid];
		out[64 * num_threads + tid] = in[64 * num_threads + tid];
		out[65 * num_threads + tid] = in[65 * num_threads + tid];
		out[66 * num_threads + tid] = in[66 * num_threads + tid];
		out[67 * num_threads + tid] = in[67 * num_threads + tid];
		out[68 * num_threads + tid] = in[68 * num_threads + tid];
		out[69 * num_threads + tid] = in[69 * num_threads + tid];
		out[70 * num_threads + tid] = in[70 * num_threads + tid];
		out[71 * num_threads + tid] = in[71 * num_threads + tid];
		out[72 * num_threads + tid] = in[72 * num_threads + tid];
		out[73 * num_threads + tid] = in[73 * num_threads + tid];
		out[74 * num_threads + tid] = in[74 * num_threads + tid];
		out[75 * num_threads + tid] = in[75 * num_threads + tid];
		out[76 * num_threads + tid] = in[76 * num_threads + tid];
		out[77 * num_threads + tid] = in[77 * num_threads + tid];
		out[78 * num_threads + tid] = in[78 * num_threads + tid];
		out[79 * num_threads + tid] = in[79 * num_threads + tid];
		out[80 * num_threads + tid] = in[80 * num_threads + tid];
		out[81 * num_threads + tid] = in[81 * num_threads + tid];
		out[82 * num_threads + tid] = in[82 * num_threads + tid];
		out[83 * num_threads + tid] = in[83 * num_threads + tid];
		out[84 * num_threads + tid] = in[84 * num_threads + tid];
		out[85 * num_threads + tid] = in[85 * num_threads + tid];
		out[86 * num_threads + tid] = in[86 * num_threads + tid];
		out[87 * num_threads + tid] = in[87 * num_threads + tid];
		out[88 * num_threads + tid] = in[88 * num_threads + tid];
		out[89 * num_threads + tid] = in[89 * num_threads + tid];
		out[90 * num_threads + tid] = in[90 * num_threads + tid];
		out[91 * num_threads + tid] = in[91 * num_threads + tid];
		out[92 * num_threads + tid] = in[92 * num_threads + tid];
		out[93 * num_threads + tid] = in[93 * num_threads + tid];
		out[94 * num_threads + tid] = in[94 * num_threads + tid];
		out[95 * num_threads + tid] = in[95 * num_threads + tid];
		out[96 * num_threads + tid] = in[96 * num_threads + tid];
		out[97 * num_threads + tid] = in[97 * num_threads + tid];
		out[98 * num_threads + tid] = in[98 * num_threads + tid];
		out[99 * num_threads + tid] = in[99 * num_threads + tid];
		out[100 * num_threads + tid] = in[100 * num_threads + tid];
		out[101 * num_threads + tid] = in[101 * num_threads + tid];
		out[102 * num_threads + tid] = in[102 * num_threads + tid];
		out[103 * num_threads + tid] = in[103 * num_threads + tid];
		out[104 * num_threads + tid] = in[104 * num_threads + tid];
		out[105 * num_threads + tid] = in[105 * num_threads + tid];
		out[106 * num_threads + tid] = in[106 * num_threads + tid];
		out[107 * num_threads + tid] = in[107 * num_threads + tid];
		out[108 * num_threads + tid] = in[108 * num_threads + tid];
		out[109 * num_threads + tid] = in[109 * num_threads + tid];
		out[110 * num_threads + tid] = in[110 * num_threads + tid];
		out[111 * num_threads + tid] = in[111 * num_threads + tid];
		out[112 * num_threads + tid] = in[112 * num_threads + tid];
		out[113 * num_threads + tid] = in[113 * num_threads + tid];
		out[114 * num_threads + tid] = in[114 * num_threads + tid];
		out[115 * num_threads + tid] = in[115 * num_threads + tid];
		out[116 * num_threads + tid] = in[116 * num_threads + tid];
		out[117 * num_threads + tid] = in[117 * num_threads + tid];
		out[118 * num_threads + tid] = in[118 * num_threads + tid];
		out[119 * num_threads + tid] = in[119 * num_threads + tid];
		out[120 * num_threads + tid] = in[120 * num_threads + tid];
		out[121 * num_threads + tid] = in[121 * num_threads + tid];
		out[122 * num_threads + tid] = in[122 * num_threads + tid];
		out[123 * num_threads + tid] = in[123 * num_threads + tid];
		out[124 * num_threads + tid] = in[124 * num_threads + tid];
		out[125 * num_threads + tid] = in[125 * num_threads + tid];
		out[126 * num_threads + tid] = in[126 * num_threads + tid];
		out[127 * num_threads + tid] = in[127 * num_threads + tid];
	}
}

