#include "hip/hip_runtime.h"
// libraries
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <omp.h>

#include <cutil_inline.h>

#include "cache_kernel.h"
#include "pm_lib.h"

// #define NUM_ITER 1

// GPU kernel error checking function
void gpu_check_error__srcpos (FILE* fp, const char* filename, size_t line)
{
  hipError_t C_E = hipGetLastError ();
  if (C_E) {
    fprintf (fp, "*** [%s:%lu] CUDA ERROR %d: %s ***\n", filename, line, C_E, 
						 hipGetErrorString (C_E));
    fflush (fp);
    exit (-1); /* abort program */
  }
}

void readConfigPower(int* wordsPerThread, int* nThreads, int* bSize, int* sensor_mask, int* sample_period, int* num_samples, char** argv)
{
	*wordsPerThread = atoi (argv[1]);
	*nThreads = atoi (argv[2]);
	*bSize = atoi (argv[3]);
	*sensor_mask = atoi (argv[4]);
	*sample_period = atoi (argv[5]);
	*num_samples = atoi (argv[6]);

}


void readConfig(int* wordsPerThread, int* nThreads, int* bSize, char** argv)
{
	*wordsPerThread = atoi (argv[1]);
	*nThreads = atoi (argv[2]);
	*bSize = atoi (argv[3]);

}

int validateResults(int nThreads, int bSize, int wordsPerThread, int* out, int* in)
{
	int i, j;
	int tmp, cnt;
	int* test;
	
	test = (int*) malloc (nThreads * sizeof (int));
	for(i = 0; i < nThreads; i++) {
		test[i] = 0;
	}

	for(i = 0; i < nThreads; i++) {
		tmp = in[i % bSize];
		for(j = 0; j < wordsPerThread; j++) {	
			tmp = in[tmp];
		}
		test[i] = tmp;
	}

	cnt = 0;
	for(i = 0; i < nThreads; i++) {
		if(test[i] != out[i]) cnt++;
	}
	
	return cnt;
}


__global__ void clear_cache (int *in)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	in[tid] = in[tid] + 1;
}

void powermonSetup (int* g_serial_fd, FILE** g_serial_fp,
                    struct termios *g_oldtio, struct termios *g_newtio,
                    char* dev, int sensor_mask, int sample_period,
                    int num_samples, int now)
{

  if(configure_tty (dev, g_serial_fd, g_serial_fp, g_oldtio, g_newtio)) {
    cleanup (*g_serial_fd, *g_serial_fp, g_oldtio);
    exit (1);
  }

  if(powermon_init (*g_serial_fd, *g_serial_fp)) {
    cleanup (*g_serial_fd, *g_serial_fp, g_oldtio);
    exit (1);
  }
  if(powermon_set_mask (sensor_mask, *g_serial_fp)) {
    cleanup (*g_serial_fd, *g_serial_fp, g_oldtio);
    exit (1);
  }
  if(powermon_set_samples (sample_period, num_samples, *g_serial_fp)) {
    cleanup (*g_serial_fd, *g_serial_fp, g_oldtio);
    exit (1);
  }

  if(powermon_set_time (now, *g_serial_fp)) {
    cleanup (*g_serial_fd, *g_serial_fp, g_oldtio);
    exit (1);
  }

}


float runTest (int* h_out, int* d_out, int* d_pchase, int nThreads, int bSize, int wordsPerThread)
{
  hipEvent_t start, stop;
	float total_time_taken;

	int num_blocks = (nThreads + bSize - 1) / bSize;

  dim3 grid (num_blocks);
  dim3 threads (bSize);

	fprintf (stderr, "number of iterations is %d\n", NUM_ITER);


  // start timer
	cutilSafeCall (hipEventCreate (&start));
	cutilSafeCall (hipEventCreate (&stop));
	cutilSafeCall (hipEventRecord (start, 0));

		for(int iter = 0; iter < NUM_ITER; iter++) {
		switch (wordsPerThread) {
			case 1:
				cache_kernel_1 <<<grid, threads>>> (nThreads, d_out, d_pchase);	
				break;
			case 2:
				cache_kernel_2 <<<grid, threads>>> (nThreads, d_out, d_pchase);	
				break;
			case 4:
				cache_kernel_4 <<<grid, threads>>> (nThreads, d_out, d_pchase);	
				break;
			case 8:
				cache_kernel_8 <<<grid, threads>>> (nThreads, d_out, d_pchase);	
				break;
			case 16:
				cache_kernel_16 <<<grid, threads>>> (nThreads, d_out, d_pchase);	
				break;
			case 32:
				cache_kernel_32 <<<grid, threads>>> (nThreads, d_out, d_pchase);	
				break;
			case 64:
				cache_kernel_64 <<<grid, threads>>> (nThreads, d_out, d_pchase);	
				break;
			case 128:
				cache_kernel_128 <<<grid, threads>>> (nThreads, d_out, d_pchase);	
				break;
			case 256:
				cache_kernel_256 <<<grid, threads>>> (nThreads, d_out, d_pchase);	
				break;
			case 512:
				cache_kernel_512 <<<grid, threads>>> (nThreads, d_out, d_pchase);	
				break;
			case 1024:
				cache_kernel_1024 <<<grid, threads>>> (nThreads, d_out, d_pchase);	
				break;
			case 2048:
				cache_kernel_2048 <<<grid, threads>>> (nThreads, d_out, d_pchase);	
				break;
			case 4096:
				cache_kernel_4096<<<grid, threads>>> (nThreads, d_out, d_pchase);	
				break;
			default:
				fprintf(stderr, "Invalid wordsPerThread: %d\n", wordsPerThread);
				total_time_taken = -1.0f;
				break;
		}
	}

  // end timer
  cutilSafeCall (hipEventRecord (stop, 0));
  cutilSafeCall (hipDeviceSynchronize ());
  cutilSafeCall (hipEventElapsedTime (&total_time_taken, start, stop));

  gpu_check_error__srcpos (stderr, __FILE__, __LINE__);

  // copy results back
  cutilSafeCall (hipMemcpy (h_out, d_out, nThreads * sizeof (int),
                             hipMemcpyDeviceToHost));
	return total_time_taken; 
}


int main(int argc, char** argv)
{
	int i;
	// long float total_dram;
	long double total_dram;

	#if MEASURE_POWER
  int g_serial_fd;
  FILE* g_serial_fp;
  struct termios g_oldtio, g_newtio;
  char* dev = "/dev/ttyUSB0";
  int sensor_mask;
  int sample_period;
  int num_samples;
	#endif

	// timer
	// hipEvent_t start, stop;
	float total_time_taken;

	// execution parameters
	int wordsPerThread;
	int nThreads;
	int bSize;

	// int num_blocks;

	// memory data structures
  int* d_out;
  int* h_out;
	int* h_pchase;
	int* d_pchase;

	#if MEASURE_POWER
  if(argc != 7) {
    fprintf(stderr, "usage: %s <word/thread> <# threads> <threads/block> <sensor mask> <sample period> <# samples>\n", 
						argv[0]);
    exit (0);
  } else {
		readConfigPower (&wordsPerThread, &nThreads, &bSize, &sensor_mask, 
								&sample_period, &num_samples, argv);
	}
	#else
  if(argc < 4) {
    fprintf(stderr, "usage: %s <word/thread> <# threads> <threads/block>\n",
						argv[0]);
    exit (0);
  } else {
		readConfig (&wordsPerThread, &nThreads, &bSize, argv);
	}

	#endif


	// find the best GPU in the system
  hipSetDevice(cutGetMaxGflopsDeviceId ());

	// clear the cache before starting the test
	#if 1
	{
		int cache_clear = 4 * 1024 * 1024; // 4 MB
		int* h_temp = (int*) malloc (cache_clear);
		for(int i = 0; i < cache_clear / sizeof (int); i++) h_temp[i] = 1;
		int* d_temp;
		cutilSafeCall (hipMalloc ((void**) &d_temp, cache_clear));
		cutilSafeCall (hipMemcpy (d_temp, h_temp, cache_clear,
															 hipMemcpyHostToDevice));

		int nt = cache_clear / sizeof (int);
		int nb = nt / 512;
		clear_cache <<<nb, 512>>> (d_temp);
  	cutilSafeCall (hipDeviceSynchronize ());
	}
	#endif


	// allocate memory
	h_out = (int*) malloc (nThreads * sizeof (int));
	h_pchase = (int*) malloc (bSize * sizeof (int));
	cutilSafeCall (hipMalloc ((void**) &d_out, nThreads * sizeof (int)));
	cutilSafeCall (hipMalloc ((void**) &d_pchase, bSize * sizeof (int)));

	// initialize memory
	for(i = 0; i < bSize - 1; i++) {
		// h_pchase[i] = i + 1;
		h_pchase[i] = i;
	}
	// h_pchase[bSize - 1] = 0;
	h_pchase[bSize - 1] = bSize - 1;
	cutilSafeCall (hipMemcpy (d_pchase, h_pchase, bSize * sizeof (int), 
														 hipMemcpyHostToDevice));

	#if MEASURE_POWER
  powermonSetup (&g_serial_fd, &g_serial_fp, &g_oldtio, &g_newtio, dev,
                 sensor_mask, sample_period, num_samples, 1);
	#endif


	#if 0
	num_blocks = (nThreads + bSize - 1) / bSize;

  dim3 grid (num_blocks);
  dim3 threads (bSize);

  // start timer
	cutilSafeCall (hipEventCreate (&start));
	cutilSafeCall (hipEventCreate (&stop));
	cutilSafeCall (hipEventRecord (start, 0));

		for(int iter = 0; iter < NUM_ITER; iter++) {
		switch (wordsPerThread) {
			case 1:
				cache_kernel_1 <<<grid, threads>>> (nThreads, d_out, d_pchase);	
				break;
			case 2:
				cache_kernel_2 <<<grid, threads>>> (nThreads, d_out, d_pchase);	
				break;
			case 4:
				cache_kernel_4 <<<grid, threads>>> (nThreads, d_out, d_pchase);	
				break;
			case 8:
				cache_kernel_8 <<<grid, threads>>> (nThreads, d_out, d_pchase);	
				break;
			case 16:
				cache_kernel_16 <<<grid, threads>>> (nThreads, d_out, d_pchase);	
				break;
			case 32:
				cache_kernel_32 <<<grid, threads>>> (nThreads, d_out, d_pchase);	
				break;
			case 64:
				cache_kernel_64 <<<grid, threads>>> (nThreads, d_out, d_pchase);	
				break;
			case 128:
				cache_kernel_128 <<<grid, threads>>> (nThreads, d_out, d_pchase);	
				break;
			case 256:
				cache_kernel_256 <<<grid, threads>>> (nThreads, d_out, d_pchase);	
				break;
			case 512:
				cache_kernel_512 <<<grid, threads>>> (nThreads, d_out, d_pchase);	
				break;
			case 1024:
				cache_kernel_1024 <<<grid, threads>>> (nThreads, d_out, d_pchase);	
				break;
			case 2048:
				cache_kernel_2048 <<<grid, threads>>> (nThreads, d_out, d_pchase);	
				break;
			case 4096:
				cache_kernel_4096<<<grid, threads>>> (nThreads, d_out, d_pchase);	
				break;
			default:
				fprintf(stderr, "Invalid wordsPerThread: %d\n", wordsPerThread);
				total_time_taken = -1.0f;
				break;
		}
	}

  // end timer
  cutilSafeCall (hipEventRecord (stop, 0));
  // cutilSafeCall (cutilDeviceSynchronize ());
  cutilSafeCall (hipDeviceSynchronize ());
  cutilSafeCall (hipEventElapsedTime (&total_time_taken, start, stop));

  gpu_check_error__srcpos (stderr, __FILE__, __LINE__);

  // copy results back
  cutilSafeCall (hipMemcpy (h_out, d_out, nThreads * sizeof (int),
                             hipMemcpyDeviceToHost));
	#endif

	#if 1	
	#if MEASURE_POWER
	#pragma omp parallel num_threads(2)
	{
		#pragma omp sections
		{
			#pragma omp section
			{
				powermon_get_samples (g_serial_fp);
				fprintf (stderr, "Power measurement code running on thread %d\n", 
								 omp_get_thread_num ());
			}
			#pragma omp section
			{
				sleep (1);
				total_time_taken = runTest (h_out, d_out, d_pchase, nThreads, bSize, 
																		wordsPerThread);
				sleep (1);
				fprintf (stderr, "Test kernel running on thread %d\n", 
								 omp_get_thread_num ());
			}
		}
	}
	#else
		total_time_taken = runTest (h_out, d_out, d_pchase, nThreads, bSize, 
																wordsPerThread);
	#endif
	#endif

	/*
	fprintf(stderr, "Results validated: %d\n", validateResults (nThreads, bSize, 
																															wordsPerThread, 
																															h_out, h_pchase));
	 */

	total_time_taken = total_time_taken / NUM_ITER;
	total_dram = ((wordsPerThread + 1) * (1.0 * nThreads/1e9)) * sizeof (int);
  fprintf (stderr, "Time taken to load %Lg GBs: %f (ms)\n", total_dram, 
					 total_time_taken);
	fprintf (stderr, "Effective bandwidth: %Lg (GB/s)\n", 
					 (total_dram/total_time_taken * 1e3));
	/*
	fprintf (stderr, "Effective compute: %lf (GFLOPS)\n", 
					 ((2.0 * wordsPerThread * nThreads)/total_time_taken)/1e6);
	 */

	free (h_out);
	cutilSafeCall (hipFree (d_out));
	free (h_pchase);
	cutilSafeCall (hipFree (d_pchase));

	return 0;
}
